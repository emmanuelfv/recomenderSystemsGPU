#include "hip/hip_runtime.h"
//  En este código se contempla la validación crusada como principal
//  característica del código

#include "validacion.h"
//-------------------------------------------------------------
//-------------------------------------------------------------
//main---------------------------------------------------------
int main(int argc, char **argv) {

#ifdef ALLOW_PRINTS
  printf("//////////////////////////////////");
  printf("//////////////////////////////////");
  printf("////////////////////////////////\n");
#endif
  
  if(argc < 7){
    printf("{prueba} {aloritmo} {dataset} {nGrupos} {nVecinos} {nRecomendaciones}\n");
    return 0;
  }
  
  int prueba = atoi(argv[1]);
  int metodo = atoi(argv[2]);
  int dataset = atoi(argv[3]);
  int kGrupos = atoi(argv[4]);
  int nVecinos = atoi(argv[5]);
  int nRecomendaciones = atoi(argv[6]);
  int parte=0;
  int similitud=0;

  if(prueba == 3 && argc < 8){
    printf("{prueba} {algoritmo} {dataset} {nGrupos} {nVecinos} {nRecomendaciones} {parte}\n");
    printf("En caso de tener metodo = 22 o 23 agregar {similitud (1,2,3,4)}\n");
    return 1;
  }
  if((metodo == ITEMKNN_SP_GPU || metodo == ITEMKNN_SP_GPU2) && argc < 9){
    printf("En caso de tener metodo = 22 o 23 agregar {similitud (1,2,3,4)}\n");
    return 1;
  }
  parte = atoi(argv[7]);
  similitud = atoi(argv[8]);
  
  #ifdef ALLOW_PRINTS
  printf("%d datos\n", vec->tREC);
  #endif

  // evaluación de tiempos
  if(prueba == 1){
    if (metodo == ITEMKNN_ORIG || metodo == ITEMKNN_OPT) {
      Eval *vec = leerDB(dataset);
      tiempos_iknn(metodo, vec, nVecinos, nRecomendaciones);
      #ifdef ALLOW_PRINTS 
      printf("Proceso completado correctamente.\n");
      #endif
      freeEval(vec);
    }
    else{
      printf("Solo se permiten los algoritmos %d %d.\n", ITEMKNN_ORIG, ITEMKNN_OPT);
    } 
      
  } 
  else if (prueba == 2){
    if (parte > kGrupos) {
      printf("La parte seleccionada de la validación cruzada es mayor al número de partes disponibles.\n");
      return 1;
    }
    if ((metodo == ITEMKNN_SP_GPU || metodo == ITEMKNN_SP_GPU2) && (similitud > 4 || similitud < 1)) {
      printf("Similitud no valida.\n");
      return 1;
    }
    if (metodo == ITEMKNN_GPU || metodo == ITEMKNN_SP || metodo == ITEMKNN_SP_GPU || metodo == ITEMKNN_SP_GPU2 || metodo == POP) {
      tiempos_exactitud(metodo, dataset, kGrupos, nVecinos, nRecomendaciones, similitud);
    }
    else {
      printf("Solo se permiten los algoritmos %d %d %d %d %d.\n", ITEMKNN_GPU, ITEMKNN_SP, ITEMKNN_SP_GPU, ITEMKNN_SP_GPU2, POP);
    }
  } 
  else if(prueba == 3){
    if (parte > kGrupos) {
      printf("La parte seleccionada de la validación cruzada es mayor al número de partes disponibles.\n");
      return 1;
    }
    if ((metodo == ITEMKNN_SP_GPU || metodo == ITEMKNN_SP_GPU2) && (similitud > 4 || similitud < 1)) {
      printf("Similitud no valida.\n");
      return 1;
    }

    if (metodo == ITEMKNN_GPU || metodo == ITEMKNN_SP || metodo == ITEMKNN_SP_GPU || metodo == ITEMKNN_SP_GPU2 || metodo == POP) {
      tiempos_exactitud_parcial(metodo, dataset, kGrupos, nVecinos, nRecomendaciones, parte, similitud);
      return 0;
    }
    else {
      printf("Solo se permiten los algoritmos %d %d %d %d %d.\n", ITEMKNN_GPU, ITEMKNN_SP, ITEMKNN_SP_GPU, ITEMKNN_SP_GPU2, POP);
      return 1;
    }
  }
  else printf("prueba no valida {1, 2, 3}");
  return 1;
}


// tiempos ikOriginal-------------------------------------------
int tiempos_iknn(int metodo, Eval *vec, int k, int n) {
  if(metodo == ITEMKNN_ORIG){
    float **R, **S, **U;
    int **rec;
    contadores(vec);
  
    R = generaMatrizR(vec);
  
    clock_t t_ini, t_fin;
    double secs;
  
    t_ini = clock();
    S = construirModelo_ikOriginal(R, k, vec->numU, vec->numI);
    t_fin = clock();
    secs = (double)(t_fin - t_ini) / CLOCKS_PER_SEC;
    printf("%.16g,", secs);
  
    t_ini = clock();
    U = allocM_F(vec->numU, vec->numI);
    rec = (int **)malloc(sizeof(int *) * vec->numU);
    for (int i = 0; i < vec->numU; i++) {
      for (int j = 0; j < vec->numI; j++)
        U[i][j] = R[j][i];
  
      rec[i] = aplicarModelo_ikOriginal(S, vec->numI, U[i], n);
    }
    t_fin = clock();
    secs = (double)(t_fin - t_ini) / CLOCKS_PER_SEC;
    printf("%.16g\n", secs);
  
  #ifdef ALLOW_PRINTS
    printf("itemKNN_m^2, resultados para usuarios 2, 5, 10\n");
    for (int i = 0; i < n; i++) {
      printf("%d %d %d\n", rec[2][i], rec[5][i], rec[10][i]);
    }
  #endif
  
    freeM_F(R, vec->numI);
    freeM_F(S, vec->numI);
    freeM_F(U, vec->numU);
    freeM_I(rec, vec->numU);
  
    return 0;
  }
  else if(metodo == ITEMKNN_OPT){

    clock_t t_ini, t_fin;
    
    // cuenta numero de ítems y de usuarios
    contadores(vec);
    int nI = vec->numI;
    int nU = vec->numU;
    
    // genera una matriz que contiene todas las evaluaciones de los usuarios
    float **R = generaMatrizR(vec);
    int **S_ind = allocM_I(nI, k);
    float **S_val = allocM_F(nI, k);
    
    // Construcción del modelo
    t_ini = clock();
    construirModelo_ik(S_ind, S_val, R, k, nU, nI);
    t_fin = clock();
    double secs = (double)(t_fin - t_ini) / CLOCKS_PER_SEC;
    printf("%.16g,", secs);
    
    // Explotación del modelo
    t_ini = clock();
    float **U = allocM_F(nU, nI);
    int **rec = (int **)malloc(sizeof(int *) * nU);
    for (int i = 0; i < nU; i++) {
      for (int j = 0; j < nI; j++) {
        U[i][j] = R[j][i];
      }
      rec[i] = aplicarModelo_ik(S_ind, S_val, nI, k, U[i], n);
    }
    t_fin = clock();
    secs = (double)(t_fin - t_ini) / CLOCKS_PER_SEC;
    printf("%.16g,", secs);
    // printf("%.16g\n", (double)(t_fin - t_ini) / CLOCKS_PER_SEC);
    
    // verificación de recomendaciones
    #ifdef ALLOW_PRINTS
    printf("itemKNN_m*K, resultados para usuarios 2, 5, 10\n");
    for (int i = 0; i < n; i++) {
      printf("%d %d %d\n", rec[2][i], rec[5][i], rec[10][i]);
    }
    #endif
    
    // Limpieza de la memoria
    freeM_F(R, nI);
    freeM_I(S_ind, nI);
    freeM_F(S_val, nI);
    freeM_F(U, nU);
    freeM_I(rec, nU);
    
    return 0;
  }
  else printf("solo se admiten metodos item KNN.");
  return 1;
}


void tiempos_exactitud(int metodo, int dataset, int kGrupos, int nVecinos, int nRecomendaciones, int similitud){

  Eval *setData;

  int i;
  Elemento *vUs, *vIt, *test;
  int *indTUs, *indTIt, *indTest, tUs, tIt;

  Elemento *Mat, *matVecinos;
  int *indSim;


  B_UI b;

  Elemento *recomendaciones;

  clock_t t_ini, t_fin;
  double secs;

  //se presentan los resultados en <resultados>
  //el orden es: tiempoConstrucción, tiempoExplotación, precisión, recall, NDCG
  float **resultados = allocM_F(kGrupos, 5);

  for(i =0; i<kGrupos; i++){

#ifdef ALLOW_PRINTS
  printf("\nvalidación: %d\n", i);
#endif

    for(int j=0; j<5; j++) resultados[i][j] = 0;

    if (metodo == ITEMKNN_GPU) {
      //lectura de dataset (partes)
      getCrossSets(dataset, kGrupos, i, &setData, &test, &indTest);
      
      #ifdef ALLOW_PRINTS
      printf("Datos: %d\n", setData->tREC);
      #endif
      
      Matrix R;
      MatrixElemento knn;
      MatrixElemento rec;
      
      //matriz de calificaciones
      R.w = tUs = setData->numU;
      R.h = tIt = setData->numI;
      // R.d = (float *) malloc(8);
      R.d = generaMatrizDatos(setData->REC, setData->tREC, tUs, tIt);
      
      //matriz de vecinos
      knn.w = nVecinos;
      knn.h = R.h;
      knn.d = (Elemento *) malloc(knn.w * knn.h * sizeof(Elemento));
      // knn.d = (Elemento *) malloc(sizeof(Elemento));
      
      //matriz de resultados
      rec.w = nRecomendaciones;
      rec.h = tUs;
      rec.d = (Elemento *) malloc(rec.w * rec.h * sizeof(Elemento));
      // rec.d = (Elemento *) malloc(sizeof(Elemento));
      
      //construcción
      t_ini = clock();  
      construccionModelo(R, knn);
      t_fin = clock();
      secs = (double)(t_fin - t_ini) / CLOCKS_PER_SEC;
      #ifdef ALLOW_PRINTS
      printf("%.16g,", secs);
      #endif
      resultados[i][0] += secs;
      
      //explotación
      t_ini = clock();
      applyModelAllGPU(R, knn, rec);
      t_fin = clock();
      secs = (double)(t_fin - t_ini) / CLOCKS_PER_SEC;
      #ifdef ALLOW_PRINTS
      printf("%.16g\n", secs);
      #endif
      resultados[i][1] += secs;
      
      //evaluación  
      evaluacion(rec.d, tUs, nRecomendaciones, test, indTest, &resultados[i][2], &resultados[i][3], &resultados[i][4]);
    
      free(R.d);
      free(knn.d);
      free(rec.d);

      freeEval(setData);
      free(test);
      free(indTest);

    }else if (metodo == ITEMKNN_SP_GPU){
      getCrossSets_SP(dataset, kGrupos, i, &vUs, &indTUs, &tUs, &vIt, &indTIt, &tIt, &test, &indTest);
      
      float *medUs, *medIt;
      getMediaPsql(indTUs, tUs, &medUs, USER);
      getMediaPsql(indTIt, tIt, &medIt, ITEM);

      get_b(vUs, vIt, indTUs, indTIt, tUs, tIt, &b);

      //Construcción del modelo
      t_ini = clock();
      construccion_SP_GPU(vUs, vIt, indTUs, indTIt, tUs, tIt, medUs, medIt, &Mat, &indSim, similitud);
      t_fin = clock();
      secs = (double)(t_fin - t_ini) / CLOCKS_PER_SEC;
      resultados[i][0] = secs;
      
      
      //Aplicacion del modelo
      t_ini = clock();
      aplicacion_SP_GPU(Mat, indSim, vUs, indTUs, vIt, indTIt, tUs, tIt, b, nVecinos, nRecomendaciones, &recomendaciones);
      t_fin = clock();
      secs = (double)(t_fin - t_ini) / CLOCKS_PER_SEC;
      resultados[i][1] = secs;

      // printf("Elementos en M: %d de %d posibles\n", indSim[tIt], tIt*tIt);
      
      //evaluación
      evaluacion(recomendaciones, tUs, nRecomendaciones, test, indTest, &resultados[i][2], &resultados[i][3], &resultados[i][4]);
      
      //liveración de memoria
      free(b.bu);
      free(b.bi);
      
      free(recomendaciones);

      free(Mat);
      free(indSim);
      
      free(medUs);
      free(medIt);

      free(test);
      free(indTest);
      free(vIt);
      free(indTIt);
      free(vUs);
      free(indTUs);
    }else if (metodo == ITEMKNN_SP_GPU2){
      getCrossSets_SP(dataset, kGrupos, i, &vUs, &indTUs, &tUs, &vIt, &indTIt, &tIt, &test, &indTest);
      
      float *medUs, *medIt;
      getMediaPsql(indTUs, tUs, &medUs, USER);
      getMediaPsql(indTIt, tIt, &medIt, ITEM);

      get_b(vUs, vIt, indTUs, indTIt, tUs, tIt, &b);


      // printf(" %d %d %f\n", tUs, tIt, b.media);
      //Construcción del modelo
      t_ini = clock();
      construccion_SP_GPU2(vUs, vIt, indTUs, indTIt, tUs, tIt, medUs, medIt, nVecinos, &matVecinos, similitud);
      t_fin = clock();
      secs = (double)(t_fin - t_ini) / CLOCKS_PER_SEC;
      resultados[i][0] = secs;
      
      //Aplicacion del modelo
      t_ini = clock();

      aplicacion_SP_GPU2(matVecinos, vUs, indTUs, tUs, tIt, b, nVecinos, nRecomendaciones, &recomendaciones);
      
      t_fin = clock();
      secs = (double)(t_fin - t_ini) / CLOCKS_PER_SEC;
      resultados[i][1] = secs;
      
      // evaluación
      evaluacion(recomendaciones, tUs, nRecomendaciones, test, indTest, &resultados[i][2], &resultados[i][3], &resultados[i][4]);
      
      //liveración de memoria
      free(b.bu);
      free(b.bi);
      
      free(recomendaciones);

      free(matVecinos);
      
      free(medUs);
      free(medIt);

      free(test);
      free(indTest);
      free(vIt);
      free(indTIt);
      free(vUs);
      free(indTUs);

    }else if (metodo == ITEMKNN_SP){
      getCrossSets_SP(dataset, kGrupos, i, &vUs, &indTUs, &tUs, &vIt, &indTIt, &tIt, &test, &indTest);
      
      float *medUs, *medIt;
      getMediaPsql(indTUs, tUs, &medUs, USER);
      getMediaPsql(indTIt, tIt, &medIt, ITEM);
      
      get_b(vUs, vIt, indTUs, indTIt, tUs, tIt, &b);
      
      
      //Construcción del modelo
      t_ini = clock();
      construccion_SP(vUs, vIt, indTUs, indTIt, tUs, tIt, medUs, medIt, &Mat, &indSim, similitud);
      t_fin = clock();
      secs = (double)(t_fin - t_ini) / CLOCKS_PER_SEC;
      resultados[i][0] = secs;
      
      //Aplicacion del modelo
      t_ini = clock();
      aplicacion_SP(Mat, indSim, vUs, indTUs, vIt, indTIt, tUs, tIt, b, nVecinos, nRecomendaciones, &recomendaciones);
      t_fin = clock();
      double secs2 = (double)(t_fin - t_ini) / CLOCKS_PER_SEC;
      resultados[i][1] = secs2;
      printf("%f, %d\n", secs + secs2, i);
      
      //evaluación
      evaluacion(recomendaciones, tUs, nRecomendaciones, test, indTest, &resultados[i][2], &resultados[i][3], &resultados[i][4]);
      
      //liveración de memoria
      free(b.bu);
      free(b.bi);
      
      free(recomendaciones);

      free(Mat);
      free(indSim);
      
      free(medUs);
      free(medIt);

      free(test);
      free(indTest);
      free(vIt);
      free(indTIt);
      free(vUs);
      free(indTUs);



    }else if(metodo == POP){
      int *pops;
      int nU;

      getCrossSets_POP(dataset, kGrupos, i, nRecomendaciones, &pops, &nU, &test, &indTest);

      simple_pop(pops, nU, nRecomendaciones, &recomendaciones);

      evaluacion(recomendaciones, nU, nRecomendaciones, test, indTest, &resultados[i][2], &resultados[i][3], &resultados[i][4]);
    }
    
#ifdef ALLOW_PRINTS
    printf("resultados%d: ", kGrupos);
    for(int j=0; j<5; j++)
      printf("%f,", resultados[i][j]);
    printf("\n");
#endif

  }

  // printf("%s ", getNombreTabla(dataset));

  float *s = (float *) malloc(sizeof(float)*5);
  for(int r=0; r<5; r++){
    s[r] = 0;
    for(i=0; i<kGrupos; i++){
      s[r] += resultados[i][r]/kGrupos;
    }
  }
  printf("%d,%d,%d,", dataset, nVecinos, similitud);
  for(i=0; i<5; i++){
    printf("%f,", s[i]);
  }
  printf("\n");

  free(s);
  freeM_F(resultados, 5);

  return;
}




void tiempos_exactitud_parcial(int metodo, int dataset, int kGrupos, int nVecinos, int nRecomendaciones, int parte, int similitud){

  Eval *setData;

  int i;
  Elemento *vUs, *vIt, *test;
  int *indTUs, *indTIt, *indTest, tUs, tIt;

  Elemento *Mat, *matVecinos;
  int *indSim;

  B_UI b;

  Elemento *recomendaciones;

  clock_t t_ini, t_fin, tt_ini, tt_fin;

  //se presentan los resultados en <resultados>
  //el orden es: tiempoConstrucción, tiempoExplotación, precisión, recall, NDCG

  tt_ini = clock();
  float *resultados = (float *) malloc(sizeof(float)*6);

  for(int j=0; j<5; j++) resultados[j] = 0;

  if (metodo == ITEMKNN_GPU) {
    //lectura de dataset (partes)
    getCrossSets(dataset, kGrupos, parte, &setData, &test, &indTest);
    
    #ifdef ALLOW_PRINTS
    printf("Datos: %d\n", setData->tREC);
    #endif
    
    Matrix R;
    MatrixElemento knn;
    MatrixElemento rec;
    
    //matriz de calificaciones
    R.w = tUs = setData->numU;
    R.h = tIt = setData->numI;
    // R.d = (float *) malloc(8);
    R.d = generaMatrizDatos(setData->REC, setData->tREC, tUs, tIt);
    
    //matriz de vecinos
    knn.w = nVecinos;
    knn.h = R.h;
    knn.d = (Elemento *) malloc(knn.w * knn.h * sizeof(Elemento));
    // knn.d = (Elemento *) malloc(sizeof(Elemento));
    
    //matriz de resultados
    rec.w = nRecomendaciones;
    rec.h = tUs;
    rec.d = (Elemento *) malloc(rec.w * rec.h * sizeof(Elemento));
    // rec.d = (Elemento *) malloc(sizeof(Elemento));
    
    //construcción
    t_ini = clock();  
    construccionModelo(R, knn);
    t_fin = clock();
    resultados[0] += (float)(t_fin - t_ini) / CLOCKS_PER_SEC;
    
    //explotación
    t_ini = clock();
    applyModelAllGPU(R, knn, rec);
    t_fin = clock();
    resultados[1] += (double)(t_fin - t_ini) / CLOCKS_PER_SEC;
    
    //evaluación  
    evaluacion(rec.d, tUs, nRecomendaciones, test, indTest, &resultados[2], &resultados[3], &resultados[4]);
  
    free(R.d);
    free(knn.d);
    free(rec.d);

    freeEval(setData);
    free(test);
    free(indTest);

  }else if (metodo == ITEMKNN_SP_GPU){
    getCrossSets_SP(dataset, kGrupos, parte, &vUs, &indTUs, &tUs, &vIt, &indTIt, &tIt, &test, &indTest);
    
    float *medUs, *medIt;
    getMediaPsql(indTUs, tUs, &medUs, USER);
    getMediaPsql(indTIt, tIt, &medIt, ITEM);

    get_b(vUs, vIt, indTUs, indTIt, tUs, tIt, &b);

    //Construcción del modelo
    t_ini = clock();
    construccion_SP_GPU(vUs, vIt, indTUs, indTIt, tUs, tIt, medUs, medIt, &Mat, &indSim, similitud);
    t_fin = clock();
    resultados[0] += (float)(t_fin - t_ini) / CLOCKS_PER_SEC;
  
    
    //Aplicacion del modelo
    t_ini = clock();
    aplicacion_SP_GPU(Mat, indSim, vUs, indTUs, vIt, indTIt, tUs, tIt, b, nVecinos, nRecomendaciones, &recomendaciones);
    t_fin = clock();
    resultados[1] += (double)(t_fin - t_ini) / CLOCKS_PER_SEC;
    
    // printf("Elementos en M: %d de %d posibles\n", indSim[tIt], tIt*tIt);
    
    //evaluación
    evaluacion(recomendaciones, tUs, nRecomendaciones, test, indTest, &resultados[2], &resultados[3], &resultados[4]);
    
    //liveración de memoria
    free(b.bu);
    free(b.bi);
    
    free(recomendaciones);

    free(Mat);
    free(indSim);
    
    free(medUs);
    free(medIt);

    free(test);
    free(indTest);
    free(vIt);
    free(indTIt);
    free(vUs);
    free(indTUs);
  }else if (metodo == ITEMKNN_SP_GPU2){
    getCrossSets_SP(dataset, kGrupos, parte, &vUs, &indTUs, &tUs, &vIt, &indTIt, &tIt, &test, &indTest);
    
    float *medUs, *medIt;
    getMediaPsql(indTUs, tUs, &medUs, USER);
    getMediaPsql(indTIt, tIt, &medIt, ITEM);

    get_b(vUs, vIt, indTUs, indTIt, tUs, tIt, &b);


    // printf(" %d %d %f\n", tUs, tIt, b.media);
    //Construcción del modelo
    t_ini = clock();
    construccion_SP_GPU2(vUs, vIt, indTUs, indTIt, tUs, tIt, medUs, medIt, nVecinos, &matVecinos, similitud);
    t_fin = clock();
    resultados[0] += (float)(t_fin - t_ini) / CLOCKS_PER_SEC;

    //Aplicacion del modelo
    t_ini = clock();
    aplicacion_SP_GPU2(matVecinos, vUs, indTUs, tUs, tIt, b, nVecinos, nRecomendaciones, &recomendaciones);
    t_fin = clock();
    resultados[1] += (double)(t_fin - t_ini) / CLOCKS_PER_SEC;
    
    // evaluación
    evaluacion(recomendaciones, tUs, nRecomendaciones, test, indTest, &resultados[2], &resultados[3], &resultados[4]);
    
    //liveración de memoria
    free(b.bu);
    free(b.bi);
    
    free(recomendaciones);

    free(matVecinos);
    
    free(medUs);
    free(medIt);

    free(test);
    free(indTest);
    free(vIt);
    free(indTIt);
    free(vUs);
    free(indTUs);

  }else if (metodo == ITEMKNN_SP){
    getCrossSets_SP(dataset, kGrupos, parte, &vUs, &indTUs, &tUs, &vIt, &indTIt, &tIt, &test, &indTest);
    
    float *medUs, *medIt;
    getMediaPsql(indTUs, tUs, &medUs, USER);
    getMediaPsql(indTIt, tIt, &medIt, ITEM);
    
    get_b(vUs, vIt, indTUs, indTIt, tUs, tIt, &b);
    
    
    //Construcción del modelo
    t_ini = clock();
    construccion_SP(vUs, vIt, indTUs, indTIt, tUs, tIt, medUs, medIt, &Mat, &indSim, similitud);
    t_fin = clock();
    resultados[0] = (float)(t_fin - t_ini) / CLOCKS_PER_SEC;;
    
    //Aplicacion del modelo
    t_ini = clock();
    aplicacion_SP(Mat, indSim, vUs, indTUs, vIt, indTIt, tUs, tIt, b, nVecinos, nRecomendaciones, &recomendaciones);
    t_fin = clock();
    resultados[1] = (float) (t_fin - t_ini) / CLOCKS_PER_SEC;
    
    //evaluación
    evaluacion(recomendaciones, tUs, nRecomendaciones, test, indTest, &resultados[2], &resultados[3], &resultados[4]);
    
    //liveración de memoria
    free(b.bu);
    free(b.bi);
    
    free(recomendaciones);

    free(Mat);
    free(indSim);
    
    free(medUs);
    free(medIt);

    free(test);
    free(indTest);
    free(vIt);
    free(indTIt);
    free(vUs);
    free(indTUs);



  }else if(metodo == POP){
    int *pops;
    int nU;

    getCrossSets_POP(dataset, kGrupos, parte, nRecomendaciones, &pops, &nU, &test, &indTest);

    simple_pop(pops, nU, nRecomendaciones, &recomendaciones);

    evaluacion(recomendaciones, nU, nRecomendaciones, test, indTest, &resultados[2], &resultados[3], &resultados[4]);
  }

  tt_fin = clock();
  resultados[5] =  (float)(tt_fin - tt_ini) / CLOCKS_PER_SEC;
    
  printf("%d,%d,%d,%d,", dataset, nVecinos, similitud, parte);
  printf("%f,", resultados[5]);
  for(i=0; i<5; i++){
    printf("%f,", resultados[i]);
  }
  printf("\n");

  free(resultados);

  return;
}
