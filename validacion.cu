#include "hip/hip_runtime.h"

//  En este código se contempla la validación crusada como principal
//  característica del código

#include "validacion.h"

//-------------------------------------------------------------
//-------------------------------------------------------------
//main---------------------------------------------------------
int main(int argc, char **argv) {

#ifdef ALLOW_PRINTS
  printf("//////////////////////////////////");
  printf("//////////////////////////////////");
  printf("////////////////////////////////\n");
#endif
  srand(time(NULL));
  
  
  if(argc < 6){
    printf("{metodo} {dataset} {nGrupos} {nVecinos} {nRecomendaciones}\n");
    return 0;
  }
  
  int metodo = atoi(argv[1]);
  int dataset = atoi(argv[2]);
  int kGrupos = atoi(argv[3]);
  int nVecinos = atoi(argv[4]);
  int nRecomendaciones = atoi(argv[5]);
  int similitud = 0;
  if(metodo == ITEMKNN_SP_GPU && argc < 7){
    printf("{metodo} {dataset} {nGrupos} {nVecinos} {nRecomendaciones}\n");
    printf("En caso de tener metodo = 22 agregar {similitud (1,2,3,4)}\n");
    return 0;
  }
  similitud = atoi(argv[6]);
  
  #ifdef ALLOW_PRINTS
  printf("%d datos\n", vec->tREC);
  #endif

  // evaluación de tiempos
  if (metodo == ITEMKNN_ORIG) {
    // evaluacion_tiempos_ikOriginal(vec, nVecinos, nRecomendaciones);
    // freeEval(vec);
    // #ifdef ALLOW_PRINTS
    // printf("Resultados correctos.\n");
    // #endif
  } else if (metodo == ITEMKNN_OPT) {
    // evaluacion_tiempos_ik(vec, nVecinos, nRecomendaciones);
    // freeEval(vec);
    // #ifdef ALLOW_PRINTS 
    // printf("Resultados correctos.\n");
    // #endif
  // } else if (0){ // (metodo == ITEMKNN_GPU){
  //   crossValidationT(metodo, vec, kGrupos, nVecinos, nRecomendaciones);
  //     // crossValidationT1(vec, kGrupos, nVecinos);
  //     freeEval(vec);

  // } else if (0){
  //   crossValidationT(metodo, vec, kGrupos, nVecinos, nRecomendaciones);
  //   // evaluacion_tiempos_ik_sp(vec, kGrupos, nVecinos);
  //   freeEval(vec);

  } else if (metodo == ITEMKNN_GPU || metodo == ITEMKNN_SP_GPU || metodo == POP) {
    evaluacion_tiempos_exactitud(metodo, dataset, kGrupos, nVecinos, nRecomendaciones, similitud);
    
  } else if (metodo == 100){
    // // return 0;
    // findB();
    // freeEval(vec);
//    popUso(dataset, nRecomendaciones)
  }

  // solo métodos ya paralelizados


  return 0;
}
//-------------------------------------------------------------
//-------------------------------------------------------------

double fEvalb(EvalR *vec, int tamVec, double *b, int tamB, 
  double mean, double l1, int nU){
  double res =0;
  for (int i =0 ; i< tamB; i++){
    res += b[i] * b[i];
  }
  res *= l1;


  for (int i =0 ; i< tamVec; i++){
    res += vec[i].rat - mean - b[vec[i].us] - b[vec[i].it + nU];
  }
  return res;
}

void findB(Eval *vec){
  int *ratings = (int*) malloc(sizeof(int) * vec->tREC);
  for(int i = 0; i< vec->tREC; i++)
    ratings[i] = (int) vec->REC[i].rat;
  
  // printf("%d\n", (int) vec->REC[i].rat);

  double mean = gsl_stats_float_mean( (float*) &vec->REC->rat, 4, vec->tREC);
  // printf("mean: %f\n", mean);

  contadores(vec);
  int tamB = vec->numU + vec->numI;
  double *b = (double *) malloc(sizeof(double)*(tamB));
  for(int i = 0; i< tamB; i++)
    b[i] = (double) (rand() % 9) - 4;

  // for(int i = 0; i< 10; i++)
  //   printf("%f\n", b[i]);

  printf("mean: %f\n", fEvalb(vec->REC, vec->tREC, b, tamB, mean, 0.015, vec->numU));

  return; 
}

//tiempos
void evaluacion_tiempos_exactitud(int metodo, int dataset, int kGrupos, int nVecinos, int nRecomendaciones, int similitud){

  Eval *setData;

  int i;
  Elemento *vUs, *vIt, *test;
  int *indTUs, *indTIt, *indTest, tUs, tIt;

  Elemento *Mat;
  int *indSim;

  // B_UI b;

  Elemento *recomendaciones;

  clock_t t_ini, t_fin;
  double secs;

  //se presentan los resultados en <resultados>
  //el orden es: tiempoConstrucción, tiempoExplotación, precisión, recall, NDCG
  float **resultados = allocM_F(kGrupos, 5);

  for(i =0; i<kGrupos; i++){

#ifdef ALLOW_PRINTS
  printf("\nvalidación: %d\n", i);
#endif

    for(int j=0; j<5; j++) resultados[i][j] = 0;

    if (metodo == ITEMKNN_GPU) {
      //lectura de dataset (partes)
      getCrossSets(dataset, kGrupos, i, &setData, &test, &indTest);
      
      #ifdef ALLOW_PRINTS
      printf("Datos: %d\n", setData->tREC);
      #endif
      
      Matrix R;
      MatrixElemento knn;
      MatrixElemento rec;
      
      //matriz de calificaciones
      R.w = tUs = setData->numU;
      R.h = tIt = setData->numI;
      // R.d = (float *) malloc(8);
      R.d = generaMatrizDatos(setData->REC, setData->tREC, tUs, tIt);
      
      //matriz de vecinos
      knn.w = nVecinos;
      knn.h = R.h;
      knn.d = (Elemento *) malloc(knn.w * knn.h * sizeof(Elemento));
      // knn.d = (Elemento *) malloc(sizeof(Elemento));
      
      //matriz de resultados
      rec.w = nRecomendaciones;
      rec.h = tUs;
      rec.d = (Elemento *) malloc(rec.w * rec.h * sizeof(Elemento));
      // rec.d = (Elemento *) malloc(sizeof(Elemento));
      
      //construcción
      t_ini = clock();  
      construccionModelo(R, knn);
      t_fin = clock();
      secs = (double)(t_fin - t_ini) / CLOCKS_PER_SEC;
      #ifdef ALLOW_PRINTS
      printf("%.16g,", secs);
      #endif
      resultados[i][0] += secs;
      
      //explotación
      t_ini = clock();
      applyModelAllGPU(R, knn, rec);
      t_fin = clock();
      secs = (double)(t_fin - t_ini) / CLOCKS_PER_SEC;
      #ifdef ALLOW_PRINTS
      printf("%.16g\n", secs);
      #endif
      resultados[i][1] += secs;
      
      //evaluación  
      evaluacion(rec.d, tUs, nRecomendaciones, test, indTest, &resultados[i][2], &resultados[i][3], &resultados[i][4]);
    
      free(R.d);
      free(knn.d);
      free(rec.d);

      freeEval(setData);
      free(test);
      free(indTest);

    }else if (metodo == ITEMKNN_SP_GPU){
      getCrossSets_SP(dataset, kGrupos, i, &vUs, &indTUs, &tUs, &vIt, &indTIt, &tIt, &test, &indTest);
      
      float *medUs, *medIt;
      getMediaPsql(indTUs, tUs, &medUs, USER);
      getMediaPsql(indTIt, tIt, &medIt, ITEM);
      
//       for(int q=0; q<tIt; q++)
//       printf("%d %f\n", q, medIt[q]);

// return;
      //Construcción del modelo
      t_ini = clock();
      construccion_SP_GPU(vUs, vIt, indTUs, indTIt, tUs, tIt, medUs, medIt, &Mat, &indSim, similitud);
      t_fin = clock();
      secs = (double)(t_fin - t_ini) / CLOCKS_PER_SEC;
      resultados[i][0] = secs;

      // for(int k=0; k<tIt; k++){
      //   printf("%d %d\n", k, indSim[k]);
      // }
      // printf("\n\n");
      // for(int k=0; k<3; k++){
      //   for(int l=indSim[100+k]; l< indSim[100+k+1]; l++){
      //     printf("%d %f\n", Mat[l].ind, Mat[l].val);
      //   }
      // }
      
      //Aplicacion del modelo
      t_ini = clock();
      aplicacion_SP_GPU(Mat, indSim, vUs, indTUs, vIt, indTIt, tUs, tIt, medIt, nVecinos, nRecomendaciones, &recomendaciones);
      t_fin = clock();
      secs = (double)(t_fin - t_ini) / CLOCKS_PER_SEC;
      resultados[i][1] = secs;

      //evaluación
      evaluacion(recomendaciones, tUs, nRecomendaciones, test, indTest, &resultados[i][2], &resultados[i][3], &resultados[i][4]);

      // free(b.bu);
      // free(b.bi);
      
      free(recomendaciones);
      
      free(Mat);
      free(indSim);
      
      free(medUs);
      free(medIt);

      free(test);
      free(indTest);
      free(vIt);
      free(indTIt);
      free(vUs);
      free(indTUs);
      // return;
      // continue;
    }else if(metodo == POP){
      int *pops;
      int nU;

      getCrossSets_POP(dataset, kGrupos, i, nRecomendaciones, &pops, &nU, &test, &indTest);

      simple_pop(pops, nU, nRecomendaciones, &recomendaciones);

      evaluacion(recomendaciones, nU, nRecomendaciones, test, indTest, &resultados[i][2], &resultados[i][3], &resultados[i][4]);
    }
    
#ifdef ALLOW_PRINTS
    printf("resultados%d: ", kGrupos);
    for(int j=0; j<5; j++)
      printf("%f,", resultados[i][j]);
    printf("\n");
#endif

  }

  printf("%s ", getNombreTabla(dataset));

  float *s = (float *) malloc(sizeof(float)*5);
  for(int r=0; r<5; r++){
    s[r] = 0;
    for(i=0; i<kGrupos; i++){
      s[r] += resultados[i][r]/kGrupos;
    }
  }
  printf("%d,%d,%d,", dataset, nVecinos, similitud);
  for(i=0; i<5; i++){
    printf("%f,", s[i]);
  }
  printf("\n");

  free(s);
  freeM_F(resultados, 5);

  return;
}




// tiempos ikOriginal-------------------------------------------
int evaluacion_tiempos_ikOriginal(Eval *vec, int k, int n) {
  float **R, **S, **U;
  int **rec;
  contadores(vec);

  R = generaMatrizR(vec);

  clock_t t_ini, t_fin;
  double secs;

  t_ini = clock();
  S = construirModelo_ikOriginal(R, k, vec->numU, vec->numI);
  t_fin = clock();
  secs = (double)(t_fin - t_ini) / CLOCKS_PER_SEC;
  printf("%.16g,", secs);

  t_ini = clock();
  U = allocM_F(vec->numU, vec->numI);
  rec = (int **)malloc(sizeof(int *) * vec->numU);
  for (int i = 0; i < vec->numU; i++) {
    for (int j = 0; j < vec->numI; j++)
      U[i][j] = R[j][i];

    rec[i] = aplicarModelo_ikOriginal(S, vec->numI, U[i], n);
  }
  t_fin = clock();
  secs = (double)(t_fin - t_ini) / CLOCKS_PER_SEC;
  printf("%.16g\n", secs);

#ifdef ALLOW_PRINTS
  printf("itemKNN_m^2, resultados para usuarios 2, 5, 10\n");
  for (int i = 0; i < n; i++) {
    printf("%d %d %d\n", rec[2][i], rec[5][i], rec[10][i]);
  }
#endif

  freeM_F(R, vec->numI);
  freeM_F(S, vec->numI);
  freeM_F(U, vec->numU);
  freeM_I(rec, vec->numU);

  return 0;
}

// tiempos ik---------------------------------------------------
int evaluacion_tiempos_ik(Eval *vec, int k, int n) {
  clock_t t_ini, t_fin;

  // cuenta numero de ítems y de usuarios
  contadores(vec);
  int nI = vec->numI;
  int nU = vec->numU;

  // genera una matriz que contiene todas las evaluaciones de los usuarios
  float **R = generaMatrizR(vec);
  int **S_ind = allocM_I(nI, k);
  float **S_val = allocM_F(nI, k);

  // Construcción del modelo
  t_ini = clock();
  construirModelo_ik(S_ind, S_val, R, k, nU, nI);
  t_fin = clock();
  double secs = (double)(t_fin - t_ini) / CLOCKS_PER_SEC;
  printf("%.16g,", secs);

  // Explotación del modelo
  t_ini = clock();
  float **U = allocM_F(nU, nI);
  int **rec = (int **)malloc(sizeof(int *) * nU);
  for (int i = 0; i < nU; i++) {
    for (int j = 0; j < nI; j++) {
      U[i][j] = R[j][i];
    }
    rec[i] = aplicarModelo_ik(S_ind, S_val, nI, k, U[i], n);
  }
  t_fin = clock();
  secs = (double)(t_fin - t_ini) / CLOCKS_PER_SEC;
  printf("%.16g,", secs);
// printf("%.16g\n", (double)(t_fin - t_ini) / CLOCKS_PER_SEC);

// verificación de recomendaciones
#ifdef ALLOW_PRINTS
  printf("itemKNN_m*K, resultados para usuarios 2, 5, 10\n");
  for (int i = 0; i < n; i++) {
    printf("%d %d %d\n", rec[2][i], rec[5][i], rec[10][i]);
  }
#endif

  // Limpieza de la memoria
  freeM_F(R, nI);
  freeM_I(S_ind, nI);
  freeM_F(S_val, nI);
  freeM_F(U, nU);
  freeM_I(rec, nU);

  return 0;
}

//tiempos ik_sp
int evaluacion_tiempos_ik_sp(Eval *vec, int kGrupos, int nVecinos){

  void crossValidationT1(Eval *vec, int kGrupos, int nVecinos);

  return 0;
}
