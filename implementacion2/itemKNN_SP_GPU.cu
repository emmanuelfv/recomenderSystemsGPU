#include "hip/hip_runtime.h"
#include "itemKNN_SP_GPU.h"
// #include <hiprand/hiprand_kernel.h>

#define checkCudaErr(ans) { gpuErrchk((ans), __FILE__, __LINE__); }
void gpuErrchk(hipError_t err, const char *file, int line){
  if (err != hipSuccess) {
    fprintf(stderr, "(error code %s %s %d)!\n", hipGetErrorString(err), file, line);
    exit(EXIT_FAILURE);
  }
  return;
}

__device__ float similitudCCP_gpu(Elemento *vecIt, int *indIt, float *medIt, int itemI, int itemJ, int h){
  // funcion coseno ajustado
  //preserentemente, se debe tener vecIt centrado en la media del usuario
  //es decir: u = [3,3,5,5]; medU = 4 -> u' = [-1,-1,1,1]
  int i=0, j=0, cont=0;
  float sum1, sum2, sum3, ri, rj;
  sum1 = sum2 = sum3 = 0;

  // printf("%d %d\n", indIt[itemI+1]-indIt[itemI], indIt[itemJ+1]-indIt[itemJ]);
  while(i<indIt[itemI+1]-indIt[itemI] && j<indIt[itemJ+1]-indIt[itemJ]){
    // printf("%d %d..\n", vecIt->REC[indIt[itemI]+i].us, vecIt->REC[indIt[itemJ]+j].us);
    if(vecIt[indIt[itemI]+i].ind == vecIt[indIt[itemJ]+j].ind){
      ri = vecIt[indIt[itemI]+i].val - medIt[itemI];
      rj = vecIt[indIt[itemJ]+j].val - medIt[itemJ];
      sum1 += ri * rj;
      sum2 += ri * ri;
      sum3 += rj * rj;  
      cont++;
      i++;
      j++;

    }
    else if(vecIt[indIt[itemI]+i].ind > vecIt[indIt[itemJ]+j].ind) j++;
    else i++;
  }
  if (sum2*sum3 == 0) return 0;

  float res = sum1/sqrt(sum2*sum3);
  if(cont < h) return res*((float) cont/h);
  return res;
}

__device__ float similitudCos_gpu(Elemento *vecIt, int *indIt, int itemI, int itemJ, int h){
  // funcion coseno ajustado
  //preserentemente, se debe tener vecIt centrado en la media del usuario
  //es decir: u = [3,3,5,5]; medU = 4 -> u' = [-1,-1,1,1]
  int i=0, j=0, cont=0;
  float sum1, sum2, sum3, ri, rj;
  sum1 = sum2 = sum3 = 0;

  // printf("%d %d\n", indIt[itemI+1]-indIt[itemI], indIt[itemJ+1]-indIt[itemJ]);
  while(i<indIt[itemI+1]-indIt[itemI] && j<indIt[itemJ+1]-indIt[itemJ]){
    // printf("%d %d..\n", vecIt->REC[indIt[itemI]+i].us, vecIt->REC[indIt[itemJ]+j].us);
    ri = vecIt[indIt[itemI]+i].val;
    rj = vecIt[indIt[itemJ]+j].val;
    if(vecIt[indIt[itemI]+i].ind == vecIt[indIt[itemJ]+j].ind){
      sum1 += ri * rj;
      sum2 += ri * ri;
      sum3 += rj * rj;  
      cont++;
      i++;
      j++;

    }
    else if(vecIt[indIt[itemI]+i].ind > vecIt[indIt[itemJ]+j].ind) {
      sum3 += rj * rj;  
      j++;
    }
    else{
      sum2 += ri * ri;
      i++;
    } 
  }
  if (sum2*sum3 == 0) return 0;

  float res = sum1/sqrt(sum2*sum3);
  if(cont < h) return res*((float) cont/h);
  return res;
}

__device__ float similitudAdCos_gpu(Elemento *vecIt, int *indIt, float *medUs, int itemI, int itemJ, int h){
  // funcion coseno ajustado
  //preserentemente, se debe tener vecIt centrado en la media del usuario
  //es decir: u = [3,3,5,5]; medU = 4 -> u' = [-1,-1,1,1]
  int i=0, j=0, cont=0;
  float sum1, sum2, sum3, ri, rj;
  sum1 = sum2 = sum3 = 0;

  // printf("%d %d\n", indIt[itemI+1]-indIt[itemI], indIt[itemJ+1]-indIt[itemJ]);
  while(i<indIt[itemI+1]-indIt[itemI] && j<indIt[itemJ+1]-indIt[itemJ]){
    // printf("%d %d..\n", vecIt->REC[indIt[itemI]+i].us, vecIt->REC[indIt[itemJ]+j].us);
    if(vecIt[indIt[itemI]+i].ind == vecIt[indIt[itemJ]+j].ind){
      ri = vecIt[indIt[itemI]+i].val - medUs[vecIt[indIt[itemI]+i].ind];
      rj = vecIt[indIt[itemJ]+j].val - medUs[vecIt[indIt[itemJ]+j].ind];
      sum1 += ri * rj;
      sum2 += ri * ri;
      sum3 += rj * rj;  
      cont++;
      i++;
      j++;
    }
    else if(vecIt[indIt[itemI]+i].ind > vecIt[indIt[itemJ]+j].ind) j++;
    else i++;
  }
  if (sum2*sum3 == 0) return 0;

  float res = sum1/sqrt(sum2*sum3);
  if(cont < h) return res*((float) cont/h);
  return res;
}

__device__ float similitudEuc_gpu(Elemento *vecIt, int *indIt, int itemI, int itemJ, int h){
  // __device__ float similitudEuc_gpu(Elemento *vecIt, int *indIt, float *medUs, int itemI, int itemJ, int h){
    // funcion coseno ajustado
  //preserentemente, se debe tener vecIt centrado en la media del usuario
  //es decir: u = [3,3,5,5]; medU = 4 -> u' = [-1,-1,1,1]
  int i=0, j=0, cont=0;
  float sum1, ri, rj;
  sum1 = 0;

  // printf("%d %d\n", indIt[itemI+1]-indIt[itemI], indIt[itemJ+1]-indIt[itemJ]);
  while(i<indIt[itemI+1]-indIt[itemI] && j<indIt[itemJ+1]-indIt[itemJ]){
    // printf("%d %d..\n", vecIt->REC[indIt[itemI]+i].us, vecIt->REC[indIt[itemJ]+j].us);
    ri = vecIt[indIt[itemI]+i].val;
    rj = vecIt[indIt[itemJ]+j].val;
    if(vecIt[indIt[itemI]+i].ind == vecIt[indIt[itemJ]+j].ind){
      
      // ri = vecIt[indIt[itemI]+i].val - medUs[vecIt[indIt[itemI]+i].ind];
      // rj = vecIt[indIt[itemJ]+j].val - medUs[vecIt[indIt[itemJ]+j].ind];
      sum1 += (ri - rj)*(ri - rj);
      cont++;
      i++;
      j++; 
    }
    else if(vecIt[indIt[itemI]+i].ind > vecIt[indIt[itemJ]+j].ind){
      sum1 += rj*rj;
      j++;
    } 
    else{
      sum1 += ri*ri;
      i++;
    } 
  }
  float res = sqrt(sum1);
  if(cont < h) return res*((float) cont/h);
  return res;
}


__global__ void construct_sp_gpu_kernel_ccp(Elemento *d_vecUs, int *d_indUs, Elemento *d_vecIt, int *d_indIt, 
  int nU, int nI, float *med, int *d_itemsConSimilitud, Elemento *d_mat, int *d_indSim){


  int i, u, us, j, it, l=0; 
  
  i = blockIdx.x * blockDim.x + threadIdx.x;
  
  if (i < nI) {

    for(u=0; u<d_indIt[i+1]-d_indIt[i]; u++){
      us = d_vecIt[d_indIt[i]+u].ind;
      for(j=0; j<d_indUs[us+1]-d_indUs[us]; j++){
        it = d_vecUs[d_indUs[us]+j].ind;
        if(i != it){
          if(d_itemsConSimilitud[i*nI + it] == 0){
            d_itemsConSimilitud[i*nI + it] = 1;
          }
        }
      }
    }
    
    for(j=0; j<nI; j++){
      if(d_itemsConSimilitud[i*nI + j]){
        float val = similitudCCP_gpu(d_vecIt, d_indIt, med, i, j, 50);
        if(val > 0.2){
          d_mat[i*nI + l].ind = j;
          d_mat[i*nI + l++].val = val;
        }
      }
    }
    d_indSim[i+1] = l;
  }
}

__global__ void construct_sp_gpu_kernel_cos(Elemento *d_vecUs, int *d_indUs, Elemento *d_vecIt, int *d_indIt, 
  int nU, int nI, int *d_itemsConSimilitud, Elemento *d_mat, int *d_indSim){


  int i, u, us, j, it, l=0; 
  int cont=0;
  
  i = blockIdx.x * blockDim.x + threadIdx.x;
  
  if (i < nI) {

    for(u=0; u<d_indIt[i+1]-d_indIt[i]; u++){
      us = d_vecIt[d_indIt[i]+u].ind;
      for(j=0; j<d_indUs[us+1]-d_indUs[us]; j++){
        it = d_vecUs[d_indUs[us]+j].ind;
        if(i != it){
          if(d_itemsConSimilitud[i*nI + it] == 0){
            d_itemsConSimilitud[i*nI + it] = 1;
            cont++;
          }
        }
      }
    }

    d_indSim[i+1] = cont;
    for(j=0; j<nI; j++){
      if(d_itemsConSimilitud[i*nI + j]){
        float simTemp = similitudCos_gpu(d_vecIt, d_indIt, i, j, 1);
        if(simTemp > 0.0){
          d_mat[i*nI + l].ind = j;
          d_mat[i*nI + l++].val = simTemp;
        }
        else d_indSim[i+1] -= 1;
      }
    }
  }
}

__global__ void construct_sp_gpu_kernel_cos_2(Elemento *d_vecUs, int *d_indUs, Elemento *d_vecIt, int *d_indIt, 
  int nU, int nI, int nVecinos, int *d_itemsConSimilitud, Elemento *d_mat){


  int i, u, us, j, k, it, pmin; 
  int cont=0;
  float min;
  
  i = blockIdx.x * blockDim.x + threadIdx.x;
  
  if (i < nI) {

    for(u=0; u<d_indIt[i+1]-d_indIt[i]; u++){
      us = d_vecIt[d_indIt[i]+u].ind;
      for(j=0; j<d_indUs[us+1]-d_indUs[us]; j++){
        it = d_vecUs[d_indUs[us]+j].ind;
        if(i != it){
          if(d_itemsConSimilitud[i*nI + it] == 0){
            d_itemsConSimilitud[i*nI + it] = 1;
            cont++;
          }
        }
      }
    }

    for(j=0; j<nI; j++){
      if(d_itemsConSimilitud[i*nI + j]){
        float simTemp = similitudCos_gpu(d_vecIt, d_indIt, i, j, 1);
        if(simTemp > 0.0){
          min = 99;
          for(k=0; k<nVecinos; k++)
          {
            if(d_mat[i*nVecinos + k].val<min){
              min = d_mat[i*nVecinos + k].val;
              pmin = k;
            }
          }
          if(simTemp>min){
            d_mat[i*nVecinos + pmin].val = simTemp;
            d_mat[i*nVecinos + pmin].ind = j;
          }
        }
      }
    }
  }
}


__global__ void construct_sp_gpu_kernel_ad_cos(Elemento *d_vecUs, int *d_indUs, Elemento *d_vecIt, int *d_indIt, 
  int nU, int nI, float *med, int *d_itemsConSimilitud, Elemento *d_mat, int *d_indSim){


  int i, u, us, j, it, l=0; 
  int cont=0;
  
  i = blockIdx.x * blockDim.x + threadIdx.x;
  
  if (i < nI) {

    for(u=0; u<d_indIt[i+1]-d_indIt[i]; u++){
      us = d_vecIt[d_indIt[i]+u].ind;
      for(j=0; j<d_indUs[us+1]-d_indUs[us]; j++){
        it = d_vecUs[d_indUs[us]+j].ind;
        if(i != it){
          if(d_itemsConSimilitud[i*nI + it] == 0){
            d_itemsConSimilitud[i*nI + it] = 1;
            cont++;
          }
        }
      }
    }

    for(j=0; j<nI; j++){
      if(d_itemsConSimilitud[i*nI + j]){
        float val = similitudAdCos_gpu(d_vecIt, d_indIt, med, i, j, 50);
        if (val > 0.2){
          d_mat[i*nI + l].ind = j;
          d_mat[i*nI + l++].val = val;
        }
      }
    }
    d_indSim[i+1] = l;
  }
}

__global__ void construct_sp_gpu_kernel_euc(Elemento *d_vecUs, int *d_indUs, Elemento *d_vecIt, int *d_indIt, 
  int nU, int nI, int *d_itemsConSimilitud, Elemento *d_mat, int *d_indSim){


  int i, u, us, j, it, l=0; 
  int cont=0;
  
  i = blockIdx.x * blockDim.x + threadIdx.x;
  
  if (i < nI) {

    for(u=0; u<d_indIt[i+1]-d_indIt[i]; u++){
      us = d_vecIt[d_indIt[i]+u].ind;
      for(j=0; j<d_indUs[us+1]-d_indUs[us]; j++){
        it = d_vecUs[d_indUs[us]+j].ind;
        if(i != it){
          if(d_itemsConSimilitud[i*nI + it] == 0){
            d_itemsConSimilitud[i*nI + it] = 1;
            cont++;
          }
        }
      }
    }

    d_indSim[i+1] = cont;
    for(j=0; j<nI; j++){
      if(d_itemsConSimilitud[i*nI + j]){
        d_mat[i*nI + l].ind = j;
        d_mat[i*nI + l++].val = similitudEuc_gpu(d_vecIt, d_indIt, i, j, 1);
      }
    }
  }
}

//
void construccion_SP_GPU(Elemento *vecUs, Elemento *vecIt, int *indUs, int *indIt, int nU, int nI, 
                float *medUs, float *medIt, Elemento **mat, int **indSim, int similitud){
  //Esta función construye el modelo en forma de una matriz dispersa con formato
  //Compresed Sparse Row CSR ya que se tiene un vector de indices que marcan el inicio
  //de cada renglón para el almacenamiento de los datos.
  
  // clock_t t_1, t_2, t_3, t_4, t_5;
  // double secs;
  
  hipError_t err = hipSuccess;
  Elemento *d_vecUs, *d_vecIt, *d_mat;
  int *d_indUs, *d_indIt, *d_indSim, *d_itemsConSimilitud;
  float *d_medUs, *d_medIt;
  size_t size;
  
  // t_1 = clock(); 
  
  //copiado en GPU
  size = sizeof(Elemento)*indUs[nU];
  err = hipMalloc(&d_vecUs, size);
  checkCudaErr(err);    
  err = hipMemcpy(d_vecUs, vecUs, size, hipMemcpyHostToDevice);
  checkCudaErr(err);
  
  size = sizeof(Elemento)*indIt[nI];
  err = hipMalloc(&d_vecIt, size);
  checkCudaErr(err);    
  err = hipMemcpy(d_vecIt, vecIt, size, hipMemcpyHostToDevice);
  checkCudaErr(err);
  
  size = sizeof(int)*(nU+1);
  err = hipMalloc(&d_indUs, size);
  checkCudaErr(err);    
  err = hipMemcpy(d_indUs, indUs, size, hipMemcpyHostToDevice);
  checkCudaErr(err);
  
  size = sizeof(int)*(nI+1);
  err = hipMalloc(&d_indIt, size);
  checkCudaErr(err);    
  err = hipMemcpy(d_indIt, indIt, size, hipMemcpyHostToDevice);
  checkCudaErr(err);
  
  size = sizeof(float)*nU;
  err = hipMalloc(&d_medUs, size);
  checkCudaErr(err);
  err = hipMemcpy(d_medUs, medUs, size, hipMemcpyHostToDevice);
  checkCudaErr(err);
  
  size = sizeof(float)*nI;
  err = hipMalloc(&d_medIt, size);
  checkCudaErr(err);
  err = hipMemcpy(d_medIt, medIt, size, hipMemcpyHostToDevice);
  checkCudaErr(err);
  
  
  size = sizeof(Elemento)*nI*nI;
  err = hipMalloc(&d_mat, size);
  checkCudaErr(err);
  
  size = sizeof(int)*nI*nI;
  err = hipMalloc(&d_itemsConSimilitud, size);
  checkCudaErr(err);
  
  size = sizeof(int)*(nI+1);
  err = hipMalloc(&d_indSim, size);
  checkCudaErr(err);
  
  // t_2 = clock();
  
  // float *med;
  if(similitud == PEARSON){
    construct_sp_gpu_kernel_ccp<<<(nI + BLOCK_SIZE_SP - 1) / BLOCK_SIZE_SP, BLOCK_SIZE_SP>>>(d_vecUs, d_indUs, 
    d_vecIt, d_indIt, nU, nI, d_medIt, d_itemsConSimilitud, d_mat, d_indSim);
  }else if(similitud == COSENO){
    construct_sp_gpu_kernel_cos<<<(nI + BLOCK_SIZE_SP - 1) / BLOCK_SIZE_SP, BLOCK_SIZE_SP>>>(d_vecUs, d_indUs, 
    d_vecIt, d_indIt, nU, nI, d_itemsConSimilitud, d_mat, d_indSim);
  }else if(similitud == COSENO_AJUSTADO){
    construct_sp_gpu_kernel_ad_cos<<<(nI + BLOCK_SIZE_SP - 1) / BLOCK_SIZE_SP, BLOCK_SIZE_SP>>>(d_vecUs, d_indUs, 
    d_vecIt, d_indIt, nU, nI, d_medUs, d_itemsConSimilitud, d_mat, d_indSim);
  }else if(similitud == EUCLIDEANO){
    construct_sp_gpu_kernel_euc<<<(nI + BLOCK_SIZE_SP - 1) / BLOCK_SIZE_SP, BLOCK_SIZE_SP>>>(d_vecUs, d_indUs, 
    d_vecIt, d_indIt, nU, nI, d_itemsConSimilitud, d_mat, d_indSim);
  }else{
    printf("Error en la selección de función objetivo.\n");
    exit(0);
  }
          
  // t_3 = clock();
  
  //  printf("%d...%d\n", nU, nI);
  mat[0] = (Elemento *) malloc(sizeof(Elemento)*nI*nI);
  indSim[0] = (int *) malloc(sizeof(int)*(1+nI));
  
  
  err = hipMemcpy(mat[0], d_mat, sizeof(Elemento)*nI*nI, hipMemcpyDeviceToHost);
  checkCudaErr(err);
  
  err = hipMemcpy(indSim[0], d_indSim, sizeof(int)*(1+nI), hipMemcpyDeviceToHost);
  checkCudaErr(err);
  // t_4 = clock();
  
  
  indSim[0][0] = 0;
  for(int i=0; i<nI; i++){
    // printf("%d_%d\n", i, indSim[0][i+1]);
    for(int j=0; j<indSim[0][i+1]; j++){
      // printf("%d_%d_%d ", indSim[0][i] + j, i*nI + j, i);
      // if(j%20==0) printf("\n");
      mat[0][indSim[0][i] + j] = mat[0][i*nI + j];
    }
    // // printf("\n");
    indSim[0][i+1] += indSim[0][i];
  }
  
  
  // // exit(1);
  // mat[0] = (Elemento *) realloc(mat[0], sizeof(Elemento)*indSim[0][nI]);
  
  hipFree(d_vecUs);
  hipFree(d_vecIt);
  hipFree(d_indUs);
  hipFree(d_indIt);
  hipFree(d_medUs);
  hipFree(d_medIt);
  hipFree(d_itemsConSimilitud);
  hipFree(d_mat);
  hipFree(d_indSim);
  // t_5 = clock();
  
  // secs = (double)(t_2 - t_1) / CLOCKS_PER_SEC;
  // printf("%.8f,", secs);
  // secs = (double)(t_3 - t_2) / CLOCKS_PER_SEC;
  // printf("%.8f,", secs);
  // secs = (double)(t_4 - t_3) / CLOCKS_PER_SEC;
  // printf("%8f,", secs);
  // secs = (double)(t_5 - t_4) / CLOCKS_PER_SEC;
  // printf("%8f,", secs);

  return;
}





__device__ int bbElemGPU(Elemento *vector, int m, int n, int val){
  //de un vector Elemento, retorna el indice de un valor espesifico
  if(n==m) return -1;
  if(n-m == 1){
    if(vector[m].ind==val) return m;
    else return -1; //no es un usuario valido
  }

  int mitad = (m + n) / 2;
  if(val<vector[mitad].ind){ 
    return bbElemGPU(vector, m, mitad, val);
  }
  else{
    return bbElemGPU(vector, mitad, n, val);
  }
}

__device__ void bbElemGPU2(Elemento *vector, int m, int n, int val, int *res){
  //de un vector Elemento, retorna el indice de un valor espesifico

  int mitad;
  while(1){
    if(n==m){
      *res = -1;
      return;
    }
    if(n-m == 1){
      *res = (vector[m].ind==val)?m:-1;
      return;
    }
    mitad = (m + n) / 2;
    if(val<vector[mitad].ind){ 
      n = mitad;
    }
    else{
      m = mitad;
    }
    
  }

}


__global__ void apply_knn_sp_gpu_kernel(Elemento *d_mat, int *d_indSim, Elemento *d_vecUs, int *d_indUs, Elemento *d_vecIt,
  int *d_indIt, int nU, int nI, float media, float *bu, float *bi, int nVecinos, Elemento *d_topVecinos, float *d_allRec, int u){

  int i, it, iit, j, k, l, pmin; 
  float min, x1;

  // hiprandState local_state;
  
  i = blockIdx.x * blockDim.x + threadIdx.x;
  
  if (i < nI) {
    it = bbElemGPU(d_vecUs, d_indUs[u], d_indUs[u+1], i);
    if(it == -1){
      //i es recomendable, se calculará rui
      
      for(j=0; j<nVecinos; j++){
        d_topVecinos[i * nVecinos + j].val = 0;
      }
      
      // Busqueda de buenos vecinos
      k=0;
      for(j=0; j<d_indUs[u+1]-d_indUs[u]; j++){
        it = d_vecUs[d_indUs[u]+j].ind;
        iit = bbElemGPU(d_mat, d_indSim[i], d_indSim[i+1], it);
        if(iit != -1){
          if(d_mat[iit].val > 0){
            // vecinos[k++] = d_mat[iit];
            k++;
            if(k>nVecinos){
              min = MAX_FLOAT;
              for(l=0; l<nVecinos; l++){
                if(d_topVecinos[i * nVecinos + l].val < min){
                  min = d_topVecinos[i * nVecinos + l].val;
                  pmin = l;
                }
              }
              if (d_mat[iit].val > min){
                d_topVecinos[i * nVecinos + pmin] = d_mat[iit];
              }
            }
            else{
              d_topVecinos[i * nVecinos + k-1] = d_mat[iit];
            }   
          }
        }
      }
      
      // delimitación de vecinos
      if(k==0) return;
      if(k>nVecinos) k = nVecinos;
      
      // calculo r_ui
      x1 = 0;
      // int x2=0;
      for(j=0; j<k; j++){
        it = bbElemGPU(d_vecUs, d_indUs[u], d_indUs[u+1], d_topVecinos[i * nVecinos + j].ind);
        
        // x1 += d_topVecinos[i * nVecinos + j].val * (d_vecUs[it].val);
        x1 += d_topVecinos[i * nVecinos + j].val * (d_vecUs[it].val - media - bu[u] - bi[d_topVecinos[i * nVecinos + j].ind]);
        // x2 += d_topVecinos[i * nVecinos + j].val;
      }
      
      //d_topRec[i].ind = i;
      //d_topRec[i].val = x1 + media + bu[u] + bi[i];
      d_allRec[u * nI + i] = x1 + media + bu[u] + bi[i];
      return;
      
    }
  }
}


__device__ void actualizaKValoresA(Elemento *vec, int k, float elem, int ind) {
  //Esta función cambia en cada llamada el peor elemento en VEC por ELEM e IND 
  //si es mejor ELEM que el peor.
      int i, pi;
  float min = MAX_FLOAT;
  
  for (i = 0; i < k; i++) {
    if (vec[i].val < min) {
      min = vec[i].val;
      pi = i;
    }
  }
  if (elem > min) {
    vec[pi].ind = ind;
    vec[pi].val = elem;
  }
  
  return;
}

__device__ void swapA(Elemento *xp, Elemento *yp) {
  //intercambio de elementos
  Elemento temp = *xp;
  *xp = *yp;
  *yp = temp;
}

__device__ void bubbleSortA(Elemento *vec, int row, int colTam) {
  //ordenamiento de elementos basico (solo para pocos elementos)
  int i, j;
  for (i = 0; i < colTam - 1; i++)
    for (j = 0; j < colTam - i - 1; j++)
      if (vec[row * colTam + j].val < vec[row * colTam + j + 1].val)
        swapA(&vec[row * colTam + j], &vec[row * colTam + j + 1]);
}


__global__ void apply_top_k_sp_gpu_kernel(int nU, int nI, int nRecomenciones, 
  float *d_topRec, Elemento *d_recomendaciones){
  
  int u = blockIdx.x * blockDim.x + threadIdx.x;
  
  if (u < nU) {
    for (int i = 0; i < nRecomenciones; i++) {
      d_recomendaciones[u * nRecomenciones + i].val = 0;
    }

    for (int i = 0; i < nI; i++)
      actualizaKValoresA(&d_recomendaciones[u * nRecomenciones], nRecomenciones, d_topRec[u * nI + i], i);

    bubbleSortA(d_recomendaciones, u, nRecomenciones);
  }
}



void aplicacion_SP_GPU(Elemento *mat, int *indSim, Elemento *vecUs, int *indUs, Elemento *vecIt,
  int *indIt, int nU, int nI, B_UI b, int nVecinos, int nRecomendaciones, Elemento **recomendaciones){
  //Esta función aplica el método esparcido mediante GPU.

  // clock_t t_1, t_2, t_3, t_4, t_5;
  // double secs;  

  hipError_t err = hipSuccess;
  Elemento *d_mat, *d_vecUs, *d_vecIt, *d_recomendaciones;
  int *d_indSim, *d_indUs, *d_indIt;
  float *d_bi, *d_bu;
  size_t size;

  // t_1 = clock();

  //copiado en GPU
  size = sizeof(Elemento)*indSim[nI];
  err = hipMalloc(&d_mat, size);
  checkCudaErr(err);
  err = hipMemcpy(d_mat, mat, size, hipMemcpyHostToDevice);
  checkCudaErr(err);
  
  size = sizeof(Elemento)*indUs[nU];
  err = hipMalloc(&d_vecUs, size);
  checkCudaErr(err);    
  err = hipMemcpy(d_vecUs, vecUs, size, hipMemcpyHostToDevice);
  checkCudaErr(err);
  
  size = sizeof(Elemento)*indIt[nI];
  err = hipMalloc(&d_vecIt, size);
  checkCudaErr(err);    
  err = hipMemcpy(d_vecIt, vecIt, size, hipMemcpyHostToDevice);
  checkCudaErr(err);
  
  size = sizeof(int)*(nI+1);
  err = hipMalloc(&d_indSim, size);
  checkCudaErr(err);    
  err = hipMemcpy(d_indSim, indSim, size, hipMemcpyHostToDevice);
  checkCudaErr(err);
  
  size = sizeof(int)*(nU+1);
  err = hipMalloc(&d_indUs, size);
  checkCudaErr(err);    
  err = hipMemcpy(d_indUs, indUs, size, hipMemcpyHostToDevice);
  checkCudaErr(err);
  
  size = sizeof(int)*(nI+1);
  err = hipMalloc(&d_indIt, size);
  checkCudaErr(err);    
  err = hipMemcpy(d_indIt, indIt, size, hipMemcpyHostToDevice);
  checkCudaErr(err);
  
  size = sizeof(float)*nI;
  err = hipMalloc(&d_bi, size);
  checkCudaErr(err);
  err = hipMemcpy(d_bi, b.bi, size, hipMemcpyHostToDevice);
  checkCudaErr(err);

  size = sizeof(float)*nU;
  err = hipMalloc(&d_bu, size);
  checkCudaErr(err);
  err = hipMemcpy(d_bu, b.bu, size, hipMemcpyHostToDevice);
  checkCudaErr(err);
  
  Elemento *d_topVecinos;//, *d_topRec;
  float *d_allRec;

  size = sizeof(Elemento)*nI*nVecinos;
  err = hipMalloc(&d_topVecinos, size);
  checkCudaErr(err);

  // size = sizeof(Elemento)*nI;
  // err = hipMalloc(&d_topRec, size);
  // checkCudaErr(err);
  
  size = sizeof(float)*nI*nU;
  err = hipMalloc(&d_allRec, size);
  checkCudaErr(err);
  
  size = sizeof(Elemento)*nRecomendaciones*nU;
  err = hipMalloc(&d_recomendaciones, size);
  checkCudaErr(err);
  
  
  recomendaciones[0] = (Elemento *) malloc(sizeof(Elemento) * nU * nRecomendaciones);
  //Elemento *rec = (Elemento *) malloc(sizeof(Elemento) * nI);
  
  int u;

  // t_2 = clock();
  // double s1=0, s2=0;

  for(u=0; u<nU; u++){
    // t_3 = clock();
    hipMemset(d_topVecinos, 0, nI*nVecinos*sizeof(Elemento));
    //hipMemset(d_topRec, 0, nRecomendaciones*sizeof(Elemento));
    
    apply_knn_sp_gpu_kernel<<<(nI + BLOCK_SIZE_SP - 1) / BLOCK_SIZE_SP, BLOCK_SIZE_SP>>>(d_mat, d_indSim, 
      d_vecUs, d_indUs, d_vecIt, d_indIt, nU, nI, b.media, d_bu, d_bi, nVecinos, d_topVecinos, d_allRec, u);
      
      
    // t_4 = clock();
    
    // size = sizeof(Elemento)*nI;
    // err = hipMemcpy(rec, d_topRec, size, hipMemcpyDeviceToHost);
    // // err = hipMemcpy(&recomendaciones[0][u*nRecomendaciones], d_recomendaciones, size, hipMemcpyDeviceToHost);
    // checkCudaErr(err);
    
    // qsort(rec, nI, sizeof(Elemento), cmpfuncElem);
    // //copiado de recomendaciones en "recomendaciones"
    // for(i=0; i<nRecomendaciones; i++){
    //   recomendaciones[0][u*nRecomendaciones + i] = rec[i];
    // }
    
    // t_5 = clock();
    
    // s1 += (double)(t_4 - t_3) / CLOCKS_PER_SEC;
    
  }
      
  
  apply_top_k_sp_gpu_kernel<<<(nU + BLOCK_SIZE_SP - 1) / BLOCK_SIZE_SP, BLOCK_SIZE_SP>>>(nU, 
    nI, nRecomendaciones, d_allRec, d_recomendaciones);
    
  size = sizeof(Elemento)*nRecomendaciones*nU;
  err = hipMemcpy(recomendaciones[0], d_recomendaciones, size, hipMemcpyDeviceToHost);
  // err = hipMemcpy(&recomendaciones[0][u*nRecomendaciones], d_recomendaciones, size, hipMemcpyDeviceToHost);
  checkCudaErr(err);
  
  // t_5 = clock();
  
  // s2 = (double)(t_5 - t_4) / CLOCKS_PER_SEC;
  // free(rec);

  hipFree(d_mat);
  hipFree(d_vecUs);
  hipFree(d_vecIt);
  hipFree(d_indSim);
  hipFree(d_indUs);
  hipFree(d_indIt);
  hipFree(d_bi);
  hipFree(d_bu);
  hipFree(d_topVecinos);
  hipFree(d_allRec);
  hipFree(d_recomendaciones);


  
  // secs = (double)(t_2 - t_1) / CLOCKS_PER_SEC;
  // printf("%8f,", secs);
  // printf("%8f,", s1);
  // printf("%8f\n", s2);

  return;
}

/*
ml100k
1693428 datos de 2825761 posibles
1715542 datos de 2829124 posibles
1712816 datos de 2829124 posibles
1726722 datos de 2829124 posibles
1710028 datos de 2829124 posibles

ml1m
10612368 datos de 13734436 posibles
10603862 datos de 13734436 posibles
10625368 datos de 13734436 posibles
10650178 datos de 13734436 posibles
10611680 datos de 13734436 posibles

amazon_0.1% 233 -> 0.25 
399,354 datos de 1,000,000 posibles
396832 datos de 1000000 posibles
397738 datos de 1000000 posibles
396401 datos de 1000000 posibles
398916 datos de 1000000 posibles

netflix_1%
997002 datos de 1000000 posibles
997002 datos de 1000000 posibles
997002 datos de 1000000 posibles
997002 datos de 1000000 posibles
997002 datos de 1000000 posibles
*/

//versión 2, más mejor
//construcción: retorna matriz elemento[nu * nVecinos];

__global__ void construct_sp_vecinos_gpu_kernel(int nI, int nVecinos, Elemento *d_mat, int *d_indSim, Elemento *d_matVecinos){

  int i, min, pk;
  i = blockIdx.x * blockDim.x + threadIdx.x;
  
  if (i < nI) {
    for (int j = 0; j < nVecinos; j++) {
      d_matVecinos[i * nVecinos + j].val = 0;
    }

    for (int j = 0; j < d_indSim[i+1] - d_indSim[i]; j++) {
      min = MAX_FLOAT; // se define como infinito
      for (int k = 0; k < nVecinos; k++) {
        if (d_matVecinos[i * nVecinos + k].val < min) {
          min = d_matVecinos[i * nVecinos + k].val;
          pk = k;
        }
      }

      if (d_mat[d_indSim[i] + j].val > min) {
        d_matVecinos[i * nVecinos + pk] = d_mat[d_indSim[i] + j];
      }
    }
  }
}


void construccion_SP_GPU2(Elemento *vecUs, Elemento *vecIt, int *indUs, int *indIt, int nU, int nI, 
  float *medUs, float *medIt, int nVecinos, Elemento **matVecinos, int similitud){
  //Esta función construye el modelo en forma de una matriz dispersa con formato
  //Compresed Sparse Row CSR ya que se tiene un vector de indices que marcan el inicio
  //de cada renglón para el almacenamiento de los datos.

  // clock_t t_1, t_2, t_3, t_4, t_5;
  // double secs;

  hipError_t err = hipSuccess;
  Elemento *d_vecUs, *d_vecIt, *d_matVecinos;
  int *d_indUs, *d_indIt, *d_itemsConSimilitud;
  float *d_medUs, *d_medIt;
  size_t size;

  // t_1 = clock(); 

  //copiado en GPU
  // printf("%d", indUs[nU]);
  size = sizeof(Elemento)*indUs[nU];
  err = hipMalloc(&d_vecUs, size);
  checkCudaErr(err);    
  err = hipMemcpy(d_vecUs, vecUs, size, hipMemcpyHostToDevice);
  checkCudaErr(err);

  size = sizeof(Elemento)*indIt[nI];
  err = hipMalloc(&d_vecIt, size);
  checkCudaErr(err);    
  err = hipMemcpy(d_vecIt, vecIt, size, hipMemcpyHostToDevice);
  checkCudaErr(err);

  size = sizeof(int)*(nU+1);
  err = hipMalloc(&d_indUs, size);
  checkCudaErr(err);    
  err = hipMemcpy(d_indUs, indUs, size, hipMemcpyHostToDevice);
  checkCudaErr(err);

  size = sizeof(int)*(nI+1);
  err = hipMalloc(&d_indIt, size);
  checkCudaErr(err);    
  err = hipMemcpy(d_indIt, indIt, size, hipMemcpyHostToDevice);
  checkCudaErr(err);

  size = sizeof(float)*nU;
  err = hipMalloc(&d_medUs, size);
  checkCudaErr(err);
  err = hipMemcpy(d_medUs, medUs, size, hipMemcpyHostToDevice);
  checkCudaErr(err);

  size = sizeof(float)*nI;
  err = hipMalloc(&d_medIt, size);
  checkCudaErr(err);
  err = hipMemcpy(d_medIt, medIt, size, hipMemcpyHostToDevice);
  checkCudaErr(err);
  
  size = sizeof(int)*nI*nI;
  err = hipMalloc(&d_itemsConSimilitud, size);
  checkCudaErr(err);

  size = sizeof(Elemento)*nI*nVecinos;
  err = hipMalloc(&d_matVecinos, size);
  checkCudaErr(err);


  // float *med;
  if(similitud == COSENO){
  // if(similitud == PEARSON){
  // construct_sp_gpu_kernel_ccp<<<(nI + BLOCK_SIZE_SP - 1) / BLOCK_SIZE_SP, BLOCK_SIZE_SP>>>(d_vecUs, d_indUs, 
  // d_vecIt, d_indIt, nU, nI, d_medIt, d_itemsConSimilitud, d_mat, d_indSim);
  // }else if(similitud == COSENO){
  construct_sp_gpu_kernel_cos_2<<<(nI + BLOCK_SIZE_SP - 1) / BLOCK_SIZE_SP, BLOCK_SIZE_SP>>>(d_vecUs, d_indUs, 
  d_vecIt, d_indIt, nU, nI, nVecinos, d_itemsConSimilitud, d_matVecinos);
  // }else if(similitud == COSENO_AJUSTADO){
  // construct_sp_gpu_kernel_ad_cos<<<(nI + BLOCK_SIZE_SP - 1) / BLOCK_SIZE_SP, BLOCK_SIZE_SP>>>(d_vecUs, d_indUs, 
  // d_vecIt, d_indIt, nU, nI, d_medUs, d_itemsConSimilitud, d_mat, d_indSim);
  // }else if(similitud == EUCLIDEANO){
  // construct_sp_gpu_kernel_euc<<<(nI + BLOCK_SIZE_SP - 1) / BLOCK_SIZE_SP, BLOCK_SIZE_SP>>>(d_vecUs, d_indUs, 
  // d_vecIt, d_indIt, nU, nI, d_itemsConSimilitud, d_mat, d_indSim);
  }else{
  printf("Error en la selección de función objetivo.\n");
  exit(0);
  }

  matVecinos[0] = (Elemento *) malloc(size);
  err = hipMemcpy(matVecinos[0], d_matVecinos, size, hipMemcpyDeviceToHost);
  checkCudaErr(err);

  hipFree(d_vecUs);
  hipFree(d_vecIt);
  hipFree(d_indUs);
  hipFree(d_indIt);
  hipFree(d_medUs);
  hipFree(d_medIt);
  hipFree(d_itemsConSimilitud);
  hipFree(d_matVecinos);

  return;
}

// __global__ getMatrixR(Elemento *vecUs, int *indUs, int nU, int nI, float *d_matR){
//   int u = blockIdx.x * blockDim.x + threadIdx.x;
  
//   if(u < nU){
//     for(int i=0; i<nI; i++) d_matR[u*nI+i] = 0;
//     for(int i=0; i<indUs[u+1]-indUs[u]; i++) d_matR[u*nI + vecUs[u+i].ind] = vecUs[u+i].val;
//   }
// }


__global__ void apply_sp_2_Kernel(Elemento *vecUs, int *indUs, int nU, int nI, int nVecinos, int nRecomendaciones, 
  float media, float *bi, float *bu, Elemento *d_matVecinos, Elemento *d_matRecomenciones){
  
  int pmin, i, j, it;
  float min, r_ui;
  int u = blockIdx.x * blockDim.x + threadIdx.x;
  if (u < nU) {
    // float min;
    for (i = 0; i < nI; i++) {
      r_ui = media + bu[u] + bi[i];
      // r_ui = 0;
      for (j = 0; j < nVecinos; j++) {
        it = bbElemGPU(vecUs, indUs[u], indUs[u+1], d_matVecinos[i * nVecinos + j].ind);
        if(it != -1){
          r_ui += d_matVecinos[i * nVecinos + j].val * (vecUs[it].val - media - bu[u] - bi[d_matVecinos[i * nVecinos + j].ind]);
          // r_ui += d_matVecinos[i * nVecinos + j].val * (vecUs[it].val);
        }
      }

      if (bbElemGPU(vecUs, indUs[u], indUs[u+1], i) != -1)
        r_ui = 0;
      
      min = 99999.9;
      for(j=0; j<nRecomendaciones; j++){
        if(d_matRecomenciones[u*nRecomendaciones + j].val < min){
          min = d_matRecomenciones[u*nRecomendaciones + j].val;
          pmin = j;
        }
      }
      if(r_ui > min){
        d_matRecomenciones[u*nRecomendaciones + pmin].val = r_ui;
        d_matRecomenciones[u*nRecomendaciones + pmin].ind = i;
      }
    }

    bubbleSortA(d_matRecomenciones, u, nRecomendaciones);
  }
}


void aplicacion_SP_GPU2(Elemento *matVecinos, Elemento *vecUs, int *indUs, int nU, int nI, 
  B_UI b, int nVecinos, int nRecomendaciones, Elemento **recomendaciones){
    
  hipError_t err = hipSuccess;
  Elemento *d_matVecinos, *d_vecUs, *d_recomendaciones;
  int *d_indUs;
  float *d_bi, *d_bu, *d_matR;
  size_t size;

  //copiado en GPU
  size = sizeof(Elemento)*nI*nVecinos;
  err = hipMalloc(&d_matVecinos, size);
  checkCudaErr(err);
  err = hipMemcpy(d_matVecinos, matVecinos, size, hipMemcpyHostToDevice);
  checkCudaErr(err);
  
  size = sizeof(Elemento)*indUs[nU];
  err = hipMalloc(&d_vecUs, size);
  checkCudaErr(err);    
  err = hipMemcpy(d_vecUs, vecUs, size, hipMemcpyHostToDevice);
  checkCudaErr(err);
  
  size = sizeof(int)*(nU+1);
  err = hipMalloc(&d_indUs, size);
  checkCudaErr(err);    
  err = hipMemcpy(d_indUs, indUs, size, hipMemcpyHostToDevice);
  checkCudaErr(err);
  
  size = sizeof(float)*nI;
  err = hipMalloc(&d_bi, size);
  checkCudaErr(err);
  err = hipMemcpy(d_bi, b.bi, size, hipMemcpyHostToDevice);
  checkCudaErr(err);

  size = sizeof(float)*nU;
  err = hipMalloc(&d_bu, size);
  checkCudaErr(err);
  err = hipMemcpy(d_bu, b.bu, size, hipMemcpyHostToDevice);
  checkCudaErr(err);

  size = sizeof(Elemento)*nRecomendaciones*nU;
  err = hipMalloc(&d_recomendaciones, size);
  checkCudaErr(err);
  err = hipMemset(d_recomendaciones, 0, size);
  checkCudaErr(err);
  
  apply_sp_2_Kernel<<<(nU + BLOCK_SIZE_SP - 1) / BLOCK_SIZE_SP, BLOCK_SIZE_SP>>>(d_vecUs, d_indUs, nU, nI, 
    nVecinos, nRecomendaciones, b.media, d_bi, d_bu, d_matVecinos, d_recomendaciones);
  
  recomendaciones[0] = (Elemento *) malloc(size);
  err = hipMemcpy(recomendaciones[0], d_recomendaciones, size, hipMemcpyDeviceToHost);
  checkCudaErr(err);
  

  hipFree(d_matVecinos);
  hipFree(d_vecUs);
  hipFree(d_indUs);
  hipFree(d_bi);
  hipFree(d_bu);
  hipFree(d_recomendaciones);

  return;
}
