#include "hip/hip_runtime.h"
#include "itemKNN_SP_GPU.h"
// #include <hiprand/hiprand_kernel.h>

void checkCudaErr(hipError_t err){
  if (err != hipSuccess) {
    fprintf(stderr, "(error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
  return;
}

__device__ float similitudCCP_gpu(Elemento *vecIt, int *indIt, float *medIt, int itemI, int itemJ, int h){
  // funcion coseno ajustado
  //preserentemente, se debe tener vecIt centrado en la media del usuario
  //es decir: u = [3,3,5,5]; medU = 4 -> u' = [-1,-1,1,1]
  int i=0, j=0, cont=0;
  float sum1, sum2, sum3, ri, rj;
  sum1 = sum2 = sum3 = 0;

  // printf("%d %d\n", indIt[itemI+1]-indIt[itemI], indIt[itemJ+1]-indIt[itemJ]);
  while(i<indIt[itemI+1]-indIt[itemI] && j<indIt[itemJ+1]-indIt[itemJ]){
    // printf("%d %d..\n", vecIt->REC[indIt[itemI]+i].us, vecIt->REC[indIt[itemJ]+j].us);
    if(vecIt[indIt[itemI]+i].ind == vecIt[indIt[itemJ]+j].ind){
      ri = vecIt[indIt[itemI]+i].val - medIt[itemI];
      rj = vecIt[indIt[itemJ]+j].val - medIt[itemJ];
      sum1 += ri * rj;
      sum2 += ri * ri;
      sum3 += rj * rj;  
      cont++;
      i++;
      j++;

    }
    else if(vecIt[indIt[itemI]+i].ind > vecIt[indIt[itemJ]+j].ind) j++;
    else i++;
  }
  if (sum2*sum3 == 0) return 0;

  float res = sum1/sqrt(sum2*sum3);
  if(cont < h) return res*((float) cont/h);
  return res;
}

__device__ float similitudCos_gpu(Elemento *vecIt, int *indIt, int itemI, int itemJ, int h){
  // funcion coseno ajustado
  //preserentemente, se debe tener vecIt centrado en la media del usuario
  //es decir: u = [3,3,5,5]; medU = 4 -> u' = [-1,-1,1,1]
  int i=0, j=0, cont=0;
  float sum1, sum2, sum3, ri, rj;
  sum1 = sum2 = sum3 = 0;

  // printf("%d %d\n", indIt[itemI+1]-indIt[itemI], indIt[itemJ+1]-indIt[itemJ]);
  while(i<indIt[itemI+1]-indIt[itemI] && j<indIt[itemJ+1]-indIt[itemJ]){
    // printf("%d %d..\n", vecIt->REC[indIt[itemI]+i].us, vecIt->REC[indIt[itemJ]+j].us);
    ri = vecIt[indIt[itemI]+i].val;
    rj = vecIt[indIt[itemJ]+j].val;
    if(vecIt[indIt[itemI]+i].ind == vecIt[indIt[itemJ]+j].ind){
      sum1 += ri * rj;
      sum2 += ri * ri;
      sum3 += rj * rj;  
      cont++;
      i++;
      j++;

    }
    else if(vecIt[indIt[itemI]+i].ind > vecIt[indIt[itemJ]+j].ind) {
      sum3 += rj * rj;  
      j++;
    }
    else{
      sum2 += ri * ri;
      i++;
    } 
  }
  if (sum2*sum3 == 0) return 0;

  float res = sum1/sqrt(sum2*sum3);
  if(cont < h) return res*((float) cont/h);
  return res;
}

__device__ float similitudAdCos_gpu(Elemento *vecIt, int *indIt, float *medUs, int itemI, int itemJ, int h){
  // funcion coseno ajustado
  //preserentemente, se debe tener vecIt centrado en la media del usuario
  //es decir: u = [3,3,5,5]; medU = 4 -> u' = [-1,-1,1,1]
  int i=0, j=0, cont=0;
  float sum1, sum2, sum3, ri, rj;
  sum1 = sum2 = sum3 = 0;

  // printf("%d %d\n", indIt[itemI+1]-indIt[itemI], indIt[itemJ+1]-indIt[itemJ]);
  while(i<indIt[itemI+1]-indIt[itemI] && j<indIt[itemJ+1]-indIt[itemJ]){
    // printf("%d %d..\n", vecIt->REC[indIt[itemI]+i].us, vecIt->REC[indIt[itemJ]+j].us);
    if(vecIt[indIt[itemI]+i].ind == vecIt[indIt[itemJ]+j].ind){
      ri = vecIt[indIt[itemI]+i].val - medUs[vecIt[indIt[itemI]+i].ind];
      rj = vecIt[indIt[itemJ]+j].val - medUs[vecIt[indIt[itemJ]+j].ind];
      sum1 += ri * rj;
      sum2 += ri * ri;
      sum3 += rj * rj;  
      cont++;
      i++;
      j++;
    }
    else if(vecIt[indIt[itemI]+i].ind > vecIt[indIt[itemJ]+j].ind) j++;
    else i++;
  }
  if (sum2*sum3 == 0) return 0;

  float res = sum1/sqrt(sum2*sum3);
  if(cont < h) return res*((float) cont/h);
  return res;
}

__device__ float similitudEuc_gpu(Elemento *vecIt, int *indIt, int itemI, int itemJ, int h){
  // __device__ float similitudEuc_gpu(Elemento *vecIt, int *indIt, float *medUs, int itemI, int itemJ, int h){
    // funcion coseno ajustado
  //preserentemente, se debe tener vecIt centrado en la media del usuario
  //es decir: u = [3,3,5,5]; medU = 4 -> u' = [-1,-1,1,1]
  int i=0, j=0, cont=0;
  float sum1, ri, rj;
  sum1 = 0;

  // printf("%d %d\n", indIt[itemI+1]-indIt[itemI], indIt[itemJ+1]-indIt[itemJ]);
  while(i<indIt[itemI+1]-indIt[itemI] && j<indIt[itemJ+1]-indIt[itemJ]){
    // printf("%d %d..\n", vecIt->REC[indIt[itemI]+i].us, vecIt->REC[indIt[itemJ]+j].us);
    ri = vecIt[indIt[itemI]+i].val;
    rj = vecIt[indIt[itemJ]+j].val;
    if(vecIt[indIt[itemI]+i].ind == vecIt[indIt[itemJ]+j].ind){
      
      // ri = vecIt[indIt[itemI]+i].val - medUs[vecIt[indIt[itemI]+i].ind];
      // rj = vecIt[indIt[itemJ]+j].val - medUs[vecIt[indIt[itemJ]+j].ind];
      sum1 += (ri - rj)*(ri - rj);
      cont++;
      i++;
      j++; 
    }
    else if(vecIt[indIt[itemI]+i].ind > vecIt[indIt[itemJ]+j].ind){
      sum1 += rj*rj;
      j++;
    } 
    else{
      sum1 += ri*ri;
      i++;
    } 
  }
  float res = sqrt(sum1);
  if(cont < h) return res*((float) cont/h);
  return res;
}


__global__ void construct_sp_gpu_kernel_ccp(Elemento *d_vecUs, int *d_indUs, Elemento *d_vecIt, int *d_indIt, 
  int nU, int nI, float *med, int *d_itemsConSimilitud, Elemento *d_mat, int *d_indSim){


  int i, u, us, j, it, l=0; 
  
  i = blockIdx.x * blockDim.x + threadIdx.x;
  
  if (i < nI) {

    for(u=0; u<d_indIt[i+1]-d_indIt[i]; u++){
      us = d_vecIt[d_indIt[i]+u].ind;
      for(j=0; j<d_indUs[us+1]-d_indUs[us]; j++){
        it = d_vecUs[d_indUs[us]+j].ind;
        if(i != it){
          if(d_itemsConSimilitud[i*nI + it] == 0){
            d_itemsConSimilitud[i*nI + it] = 1;
          }
        }
      }
    }
    
    for(j=0; j<nI; j++){
      if(d_itemsConSimilitud[i*nI + j]){
        float val = similitudCCP_gpu(d_vecIt, d_indIt, med, i, j, 50);
        if(val > 0.2){
          d_mat[i*nI + l].ind = j;
          d_mat[i*nI + l++].val = val;
        }
      }
    }
    d_indSim[i+1] = l;
  }
}

__global__ void construct_sp_gpu_kernel_cos(Elemento *d_vecUs, int *d_indUs, Elemento *d_vecIt, int *d_indIt, 
  int nU, int nI, int *d_itemsConSimilitud, Elemento *d_mat, int *d_indSim){


  int i, u, us, j, it, l=0; 
  int cont=0;
  
  i = blockIdx.x * blockDim.x + threadIdx.x;
  
  if (i < nI) {

    for(u=0; u<d_indIt[i+1]-d_indIt[i]; u++){
      us = d_vecIt[d_indIt[i]+u].ind;
      for(j=0; j<d_indUs[us+1]-d_indUs[us]; j++){
        it = d_vecUs[d_indUs[us]+j].ind;
        if(i != it){
          if(d_itemsConSimilitud[i*nI + it] == 0){
            d_itemsConSimilitud[i*nI + it] = 1;
            cont++;
          }
        }
      }
    }

    d_indSim[i+1] = cont;
    for(j=0; j<nI; j++){
      if(d_itemsConSimilitud[i*nI + j]){
        d_mat[i*nI + l].ind = j;
        d_mat[i*nI + l++].val = similitudCos_gpu(d_vecIt, d_indIt, i, j, 1);
      }
    }
  }
}


__global__ void construct_sp_gpu_kernel_ad_cos(Elemento *d_vecUs, int *d_indUs, Elemento *d_vecIt, int *d_indIt, 
  int nU, int nI, float *med, int *d_itemsConSimilitud, Elemento *d_mat, int *d_indSim){


  int i, u, us, j, it, l=0; 
  int cont=0;
  
  i = blockIdx.x * blockDim.x + threadIdx.x;
  
  if (i < nI) {

    for(u=0; u<d_indIt[i+1]-d_indIt[i]; u++){
      us = d_vecIt[d_indIt[i]+u].ind;
      for(j=0; j<d_indUs[us+1]-d_indUs[us]; j++){
        it = d_vecUs[d_indUs[us]+j].ind;
        if(i != it){
          if(d_itemsConSimilitud[i*nI + it] == 0){
            d_itemsConSimilitud[i*nI + it] = 1;
            cont++;
          }
        }
      }
    }

    for(j=0; j<nI; j++){
      if(d_itemsConSimilitud[i*nI + j]){
        float val = similitudAdCos_gpu(d_vecIt, d_indIt, med, i, j, 50);
        if (val > 0.2){
          d_mat[i*nI + l].ind = j;
          d_mat[i*nI + l++].val = val;
        }
      }
    }
    d_indSim[i+1] = l;
  }
}

__global__ void construct_sp_gpu_kernel_euc(Elemento *d_vecUs, int *d_indUs, Elemento *d_vecIt, int *d_indIt, 
  int nU, int nI, int *d_itemsConSimilitud, Elemento *d_mat, int *d_indSim){


  int i, u, us, j, it, l=0; 
  int cont=0;
  
  i = blockIdx.x * blockDim.x + threadIdx.x;
  
  if (i < nI) {

    for(u=0; u<d_indIt[i+1]-d_indIt[i]; u++){
      us = d_vecIt[d_indIt[i]+u].ind;
      for(j=0; j<d_indUs[us+1]-d_indUs[us]; j++){
        it = d_vecUs[d_indUs[us]+j].ind;
        if(i != it){
          if(d_itemsConSimilitud[i*nI + it] == 0){
            d_itemsConSimilitud[i*nI + it] = 1;
            cont++;
          }
        }
      }
    }

    d_indSim[i+1] = cont;
    for(j=0; j<nI; j++){
      if(d_itemsConSimilitud[i*nI + j]){
        d_mat[i*nI + l].ind = j;
        d_mat[i*nI + l++].val = similitudEuc_gpu(d_vecIt, d_indIt, i, j, 1);
      }
    }
  }
}

//
void construccion_SP_GPU(Elemento *vecUs, Elemento *vecIt, int *indUs, int *indIt, int nU, int nI, 
                float *medUs, float *medIt, Elemento **mat, int **indSim, int similitud){
  //Esta función construye el modelo en forma de una matriz dispersa con formato
  //Compresed Sparse Row CSR ya que se tiene un vector de indices que marcan el inicio
  //de cada renglón para el almacenamiento de los datos.
  
  // clock_t t_1, t_2, t_3, t_4, t_5;
  // double secs;
  // t_1 = clock(); 
  // t_2 = clock();
  // t_3 = clock();
  // t_4 = clock();
  // t_5 = clock();

  hipError_t err = hipSuccess;
  Elemento *d_vecUs, *d_vecIt, *d_mat;
  int *d_indUs, *d_indIt, *d_indSim, *d_itemsConSimilitud;
  float *d_medUs, *d_medIt;
  size_t size;

  
  //copiado en GPU
  size = sizeof(Elemento)*indUs[nU];
  err = hipMalloc(&d_vecUs, size);
  checkCudaErr(err);    
  err = hipMemcpy(d_vecUs, vecUs, size, hipMemcpyHostToDevice);
  checkCudaErr(err);
  
  size = sizeof(Elemento)*indIt[nI];
  err = hipMalloc(&d_vecIt, size);
  checkCudaErr(err);    
  err = hipMemcpy(d_vecIt, vecIt, size, hipMemcpyHostToDevice);
  checkCudaErr(err);
  
  size = sizeof(int)*(nU+1);
  err = hipMalloc(&d_indUs, size);
  checkCudaErr(err);    
  err = hipMemcpy(d_indUs, indUs, size, hipMemcpyHostToDevice);
  checkCudaErr(err);
  
  size = sizeof(int)*(nI+1);
  err = hipMalloc(&d_indIt, size);
  checkCudaErr(err);    
  err = hipMemcpy(d_indIt, indIt, size, hipMemcpyHostToDevice);
  checkCudaErr(err);
  
  size = sizeof(float)*nU;
  err = hipMalloc(&d_medUs, size);
  checkCudaErr(err);
  err = hipMemcpy(d_medUs, medUs, size, hipMemcpyHostToDevice);
  checkCudaErr(err);
  
  size = sizeof(float)*nI;
  err = hipMalloc(&d_medIt, size);
  checkCudaErr(err);
  err = hipMemcpy(d_medIt, medIt, size, hipMemcpyHostToDevice);
  checkCudaErr(err);
  
  
  size = sizeof(Elemento)*nI*nI;
  err = hipMalloc(&d_mat, size);
  checkCudaErr(err);
  
  size = sizeof(int)*nI*nI;
  err = hipMalloc(&d_itemsConSimilitud, size);
  checkCudaErr(err);
  
  size = sizeof(int)*(nI+1);
  err = hipMalloc(&d_indSim, size);
  checkCudaErr(err);
  
  
  // float *med;
  if(similitud == PEARSON){
    construct_sp_gpu_kernel_ccp<<<(nI + BLOCK_SIZE_SP - 1) / BLOCK_SIZE_SP, BLOCK_SIZE_SP>>>(d_vecUs, d_indUs, 
                                      d_vecIt, d_indIt, nU, nI, d_medIt, d_itemsConSimilitud, d_mat, d_indSim);
  }else if(similitud == COSENO){
    construct_sp_gpu_kernel_cos<<<(nI + BLOCK_SIZE_SP - 1) / BLOCK_SIZE_SP, BLOCK_SIZE_SP>>>(d_vecUs, d_indUs, 
                                              d_vecIt, d_indIt, nU, nI, d_itemsConSimilitud, d_mat, d_indSim);
  }else if(similitud == COSENO_AJUSTADO){
    construct_sp_gpu_kernel_ad_cos<<<(nI + BLOCK_SIZE_SP - 1) / BLOCK_SIZE_SP, BLOCK_SIZE_SP>>>(d_vecUs, d_indUs, 
                                    d_vecIt, d_indIt, nU, nI, d_medUs, d_itemsConSimilitud, d_mat, d_indSim);
  }else if(similitud == EUCLIDEANO){
  construct_sp_gpu_kernel_euc<<<(nI + BLOCK_SIZE_SP - 1) / BLOCK_SIZE_SP, BLOCK_SIZE_SP>>>(d_vecUs, d_indUs, 
                                            d_vecIt, d_indIt, nU, nI, d_itemsConSimilitud, d_mat, d_indSim);
  }else{
    printf("Error en la selección de función objetivo.\n");
    exit(0);
  }
    
    
    //  printf("%d...%d\n", nU, nI);
    mat[0] = (Elemento *) malloc(sizeof(Elemento)*nI*nI);
    indSim[0] = (int *) malloc(sizeof(int)*(1+nI));
    
    
    err = hipMemcpy(mat[0], d_mat, sizeof(Elemento)*nI*nI, hipMemcpyDeviceToHost);
    checkCudaErr(err);
    
  err = hipMemcpy(indSim[0], d_indSim, sizeof(int)*(1+nI), hipMemcpyDeviceToHost);
  checkCudaErr(err);
  

  indSim[0][0] = 0;
  for(int i=0; i<nI; i++){
    // printf("%d_%d\n", i, indSim[0][i+1]);
    for(int j=0; j<indSim[0][i+1]; j++){
      // printf("%d_%d_%d ", indSim[0][i] + j, i*nI + j, i);
      // if(j%20==0) printf("\n");
      mat[0][indSim[0][i] + j] = mat[0][i*nI + j];
    }
    // // printf("\n");
    indSim[0][i+1] += indSim[0][i];
  }


  // // exit(1);
  // mat[0] = (Elemento *) realloc(mat[0], sizeof(Elemento)*indSim[0][nI]);

  hipFree(d_vecUs);
  hipFree(d_vecIt);
  hipFree(d_indUs);
  hipFree(d_indIt);
  hipFree(d_medUs);
  hipFree(d_medIt);
  hipFree(d_itemsConSimilitud);
  hipFree(d_mat);
  hipFree(d_indSim);

  // secs = (double)(t_2 - t_1) / CLOCKS_PER_SEC;
  // printf("%.16g,", secs);
  // secs = (double)(t_3 - t_2) / CLOCKS_PER_SEC;
  // printf("%.16g,", secs);
  // secs = (double)(t_4 - t_3) / CLOCKS_PER_SEC;
  // printf("%.16g,", secs);
  // secs = (double)(t_5 - t_4) / CLOCKS_PER_SEC;
  // printf("%.16g\n", secs);

  return;
}





__device__ int bbElemGPU(Elemento *vector, int m, int n, int val){
  //de un vector Elemento, retorna el indice de un valor espesifico
  if(n-m <= 1){
    if(vector[m].ind==val) return m;
    else return -1; //no es un usuario valido
  }

  int mitad = (m + n) / 2;
  if(val<vector[mitad].ind){ 
    return bbElemGPU(vector, m, mitad, val);
  }
  else{
    return bbElemGPU(vector, mitad, n, val);
  }
}


__global__ void apply_knn_sp_gpu_kernel(Elemento *d_mat, int *d_indSim, Elemento *d_vecUs, int *d_indUs, Elemento *d_vecIt,
  int *d_indIt, int nU, int nI, float *d_medIt, int nVecinos, int nRecomendaciones, Elemento *d_recomendaciones, 
  Elemento *d_topVecinos, Elemento *d_topRec, int u){

  int i, it, iit, j, k, l, pmin; 
  float min, x1;

  // hiprandState local_state;
  
  i = blockIdx.x * blockDim.x + threadIdx.x;
  
  if (i < nI) {
    it = bbElemGPU(d_vecUs, d_indUs[u], d_indUs[u+1], i);
    if(it == -1){
      //i es recomendable, se calculará rui
      
      for(j=0; j<nVecinos; j++){
        d_topVecinos[i * nVecinos + j].val = 0;
      }
      
      // Busqueda de buenos vecinos
      k=0;
      for(j=0; j<d_indUs[u+1]-d_indUs[u]; j++){
        it = d_vecUs[d_indUs[u]+j].ind;
        iit = bbElemGPU(d_mat, d_indSim[i], d_indSim[i+1], it);
        if(iit != -1){
          if(d_mat[iit].val > 0){
            // vecinos[k++] = d_mat[iit];
            k++;
            if(k>nVecinos){
              min = MAX_FLOAT;
              for(l=0; l<nVecinos; l++){
                if(d_topVecinos[i * nVecinos + l].val < min){
                  min = d_topVecinos[i * nVecinos + l].val;
                  pmin = l;
                }
              }
              if (d_mat[iit].val > min){
                d_topVecinos[i * nVecinos + pmin] = d_mat[iit];
              }
            }
            else{
              d_topVecinos[i * nVecinos + k-1] = d_mat[iit];
            }   
          }
        }
      }
      
      // delimitación de vecinos
      if(k==0) return;
      if(k>nVecinos) k = nVecinos;
      
      // calculo r_ui
      x1 = 0;
      // int x2=0;
      for(j=0; j<k; j++){
        it = bbElemGPU(d_vecUs, d_indUs[u], d_indUs[u+1], d_topVecinos[i * nVecinos + j].ind);
        
        // x1 += d_topVecinos[i * nVecinos + j].val * (d_vecUs[it].val);
        x1 += d_topVecinos[i * nVecinos + j].val * (d_vecUs[it].val - d_medIt[it]);
        // x2 += d_topVecinos[i * nVecinos + j].val;
      }
      
      d_topRec[i].ind = i;
           d_topRec[i].val = d_medIt[i] + x1;
      // d_topRec[i].val = hiprand(&local_state);
      // d_topRec[i].val = x1;
      
    }
  }
}



void aplicacion_SP_GPU(Elemento *mat, int *indSim, Elemento *vecUs, int *indUs, Elemento *vecIt,
  int *indIt, int nU, int nI, float *medIt, int nVecinos, int nRecomendaciones, Elemento **recomendaciones){
  //Esta función aplica el método esparcido mediante GPU.

  hipError_t err = hipSuccess;
  Elemento *d_mat, *d_vecUs, *d_vecIt, *d_recomendaciones;
  int *d_indSim, *d_indUs, *d_indIt;
  float *d_medIt;
  size_t size;

  //copiado en GPU
  size = sizeof(Elemento)*indSim[nI];
  err = hipMalloc(&d_mat, size);
  checkCudaErr(err);
  err = hipMemcpy(d_mat, mat, size, hipMemcpyHostToDevice);
  checkCudaErr(err);
  
  size = sizeof(Elemento)*indUs[nU];
  err = hipMalloc(&d_vecUs, size);
  checkCudaErr(err);    
  err = hipMemcpy(d_vecUs, vecUs, size, hipMemcpyHostToDevice);
  checkCudaErr(err);
  
  size = sizeof(Elemento)*indIt[nI];
  err = hipMalloc(&d_vecIt, size);
  checkCudaErr(err);    
  err = hipMemcpy(d_vecIt, vecIt, size, hipMemcpyHostToDevice);
  checkCudaErr(err);
  
  size = sizeof(int)*(nI+1);
  err = hipMalloc(&d_indSim, size);
  checkCudaErr(err);    
  err = hipMemcpy(d_indSim, indSim, size, hipMemcpyHostToDevice);
  checkCudaErr(err);
  
  size = sizeof(int)*(nU+1);
  err = hipMalloc(&d_indUs, size);
  checkCudaErr(err);    
  err = hipMemcpy(d_indUs, indUs, size, hipMemcpyHostToDevice);
  checkCudaErr(err);
  
  size = sizeof(int)*(nI+1);
  err = hipMalloc(&d_indIt, size);
  checkCudaErr(err);    
  err = hipMemcpy(d_indIt, indIt, size, hipMemcpyHostToDevice);
  checkCudaErr(err);
  
  size = sizeof(float)*nI;
  err = hipMalloc(&d_medIt, size);
  checkCudaErr(err);
  err = hipMemcpy(d_medIt, medIt, size, hipMemcpyHostToDevice);
  checkCudaErr(err);
  
  Elemento *d_topVecinos, *d_topRec;
  
  size = sizeof(Elemento)*nI*nVecinos;
  err = hipMalloc(&d_topVecinos, size);
  checkCudaErr(err);
  
  size = sizeof(Elemento)*nI;
  err = hipMalloc(&d_topRec, size);
  checkCudaErr(err);
  
  recomendaciones[0] = (Elemento *) malloc(sizeof(Elemento) * nU * nRecomendaciones);
  Elemento *rec = (Elemento *) malloc(sizeof(Elemento) * nI);

  int u, i;

  for(u=0; u<nU; u++){
    hipMemset(d_topVecinos, 0, nI*nVecinos*sizeof(Elemento));
    hipMemset(d_topRec, 0, nRecomendaciones*sizeof(Elemento));
    
    // printf("ok10 %d\n", u);
  
    apply_knn_sp_gpu_kernel<<<(nI + BLOCK_SIZE_SP - 1) / BLOCK_SIZE_SP, BLOCK_SIZE_SP>>>(d_mat, d_indSim, 
      d_vecUs, d_indUs, d_vecIt, d_indIt, nU, nI, d_medIt, nVecinos, nRecomendaciones, d_recomendaciones, 
      d_topVecinos, d_topRec, u);

      // printf("ok11 %d\n", u);

    size = sizeof(Elemento)*nI;
    err = hipMemcpy(rec, d_topRec, size, hipMemcpyDeviceToHost);
    // err = hipMemcpy(&recomendaciones[0][u*nRecomendaciones], d_recomendaciones, size, hipMemcpyDeviceToHost);
    checkCudaErr(err);

    // printf("ok12 %d\n", u);

    qsort(rec, nI, sizeof(Elemento), cmpfuncElem);
    //copiado de recomendaciones en "recomendaciones"
    // rui = 0;
    for(i=0; i<nRecomendaciones; i++){
      recomendaciones[0][u*nRecomendaciones + i] = rec[i];
      // rui += recomendaciones[0][u*nRecomendaciones + i].val;
    }


  }

  free(rec);

  hipFree(d_mat);
  hipFree(d_vecUs);
  hipFree(d_vecIt);
  hipFree(d_indSim);
  hipFree(d_indUs);
  hipFree(d_indIt);
  hipFree(d_medIt);
  hipFree(d_topVecinos);
  hipFree(d_topRec);

  return;
}

/*
ml100k
1693428 datos de 2825761 posibles
1715542 datos de 2829124 posibles
1712816 datos de 2829124 posibles
1726722 datos de 2829124 posibles
1710028 datos de 2829124 posibles

ml1m
10612368 datos de 13734436 posibles
10603862 datos de 13734436 posibles
10625368 datos de 13734436 posibles
10650178 datos de 13734436 posibles
10611680 datos de 13734436 posibles

amazon_0.1% 233 -> 0.25 
399,354 datos de 1,000,000 posibles
396832 datos de 1000000 posibles
397738 datos de 1000000 posibles
396401 datos de 1000000 posibles
398916 datos de 1000000 posibles

si m=1000: 

4*m*m = 4 megas

960

12*960*m = 12 megas



8 * m*m*densidad + m*4 | desidad [0,1]

dispersión valida: 0.1

1
1
1
0
0 m=5
1
0
0
0
0
1
1
0

Enviar borrador lunes mañana

gcc -DDEBUG -DALLOW_PRINTS 

#ifdef DEBUG
  checkMemory(ptr);
#endif

2, 0.4 <- i0
4, 0.8 <- i2
5, 0.5 <- i3

0 <- i0
2 <- i1
2 <- i2
3

*/

