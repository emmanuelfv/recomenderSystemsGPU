#include "hip/hip_runtime.h"

/*
Verción derivada de la version 6
La intención es reducir el tamaño del código escrito en esta etapa,
actualemente existen 908 lineas de código, el uso de las librerías pertinentes 
deberían ser capaces de reducir el tamaño en esta implementación.
*/

#include <cstdlib>
#include <hip/hip_runtime.h>

#include "itemKNN_GPU.h"


//#define ALLOW_PRINTS


void printM(const Matrix M) {
  // muestra la matriz en forma cuadrada
  for (int i = 0; i < M.h; i++) {
    for (int j = 0; j < M.w; j++) {
      printf("%5.0f ", M.d[i * M.w + j]);
    }
    printf("\n");
  }
  printf("\n");
  return;
}

float *generaMatrizDatos(EvalR *train, int trainTam, int numUs, int numIt) {
  //genera el vector de la matriz R
  float *R;
  R = (float *) malloc(numUs * numIt * sizeof(float));
  if(R== NULL){
    printf("Error NULL en R\n");
    exit(1);
  }
  for (int i = 0; i < numUs * numIt; i++) {
    R[i] = 0;
  }
  
  for (int i = 0; i < trainTam; i++) {
    if ((train[i].it) * numUs + (train[i].us) >= numUs * numIt){
      printf("Error, almacenando %d %d %d %d.", i, train[i].us, train[i].it, (train[i].it) * numUs + (train[i].us));
      exit(1);
    }
    R[(train[i].it) * numUs + (train[i].us)] = train[i].rat;
  }

  return R;
}


float *generaDatosR(Eval *dat) {
  // genera el vector de la matriz (misma función que "generaMatrizDatos()")
  float *R;
  R = (float *)malloc(sizeof(float) * dat->numU * dat->numI);
  for (int i = 0; i < dat->numI * dat->numU; i++) {
    R[i] = 0;
  }
  
  for (int i = 0; i < dat->tREC; i++) {
    R[(dat->REC[i].it) * dat->numU + (dat->REC[i].us)] = dat->REC[i].rat;
  }
  
  return R;
}


__global__ void MatSimilitudKernel(Matrix R, Matrix S) {
  //Genera el computo de la similitud entre elementos (ítems)
  float a = 0, b = 0, c = 0;
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  if (row < S.h && col < S.w) {
    for (int i = 0; i < R.w; i++) {
      a += R.d[row * R.w + i] * R.d[col * R.w + i];
      b += R.d[row * R.w + i] * R.d[row * R.w + i];
      c += R.d[col * R.w + i] * R.d[col * R.w + i];
    }
    float div = sqrt(b * c);
    div==0? S.d[row * S.w + col] = 0 : S.d[row * S.w + col] = a / div;
    if (col == row) 
      S.d[row * S.w + col] = 0;
  }
}

__global__ void MatVecinosKernel(Matrix S, MatrixElemento knn) {
  //Genera a partir de una matriz de similitudes una de vecinos
  int row, pj;
  row = blockIdx.x * blockDim.x + threadIdx.x;
  float min;
  
  if (row < S.h) {
    for (int i = 0; i < knn.w; i++) {
      knn.d[row * knn.w + i].val = 0;
    }
    for (int i = 0; i < S.w; i++) {
      min = MAX_FLOAT; // se define como infinito
      for (int j = 0; j < knn.w; j++) {
        if (knn.d[row * knn.w + j].val < min) {
          min = knn.d[row * knn.w + j].val;
          pj = j;
        }
      }
      if (S.d[row * S.w + i] > min) {
        knn.d[row * knn.w + pj].val = S.d[row * S.w + i];
        knn.d[row * knn.w + pj].ind = i;
      }
    }
  }
}

void construccionModelo(Matrix R, MatrixElemento knn) {
  //obtiene una matriz de vecinos cercanos para cada ítem
  Matrix d_R;
  d_R.w = R.w; //nU
  d_R.h = R.h; //nI
  size_t size = R.w * R.h * sizeof(float);
  hipMalloc(&d_R.d, size);
  hipMemcpy(d_R.d, R.d, size, hipMemcpyHostToDevice);
  
  Matrix d_S;
  d_S.w = R.h; //nI
  d_S.h = R.h; //nI
  size = d_S.w * d_S.h * sizeof(float);
  hipMalloc(&d_S.d, size);
  
  dim3 dimBlock(BLOCK_SIZE_IK, BLOCK_SIZE_IK);
  dim3 dimGrid((d_S.w + dimBlock.x - 1) / dimBlock.x, 
  (d_S.h + dimBlock.y - 1) / dimBlock.y);
  MatSimilitudKernel<<<dimGrid, dimBlock>>>(d_R, d_S);

  MatrixElemento d_knn;
  d_knn.w = knn.w; //nV
  d_knn.h = knn.h; //nI
  size = knn.w * knn.h * sizeof(Elemento);
  
  hipMalloc(&d_knn.d, size);

  MatVecinosKernel<<<(d_knn.h + BLOCK_SIZE_IK - 1) / BLOCK_SIZE_IK, BLOCK_SIZE_IK>>>(d_S, d_knn);
    
  hipMemcpy(knn.d, d_knn.d, size, hipMemcpyDeviceToHost);
  
  hipFree(d_R.d);
  hipFree(d_S.d);
  hipFree(d_knn.d);
  
  return;
}
    
__device__ void actualizaKValores(Elemento *vec, int k, float elem, int ind) {
  //Esta función cambia en cada llamada el peor elemento en VEC por ELEM e IND 
  //si es mejor ELEM que el peor.
      int i, pi;
  float min = MAX_FLOAT;
  
  for (i = 0; i < k; i++) {
    if (vec[i].val < min) {
      min = vec[i].val;
      pi = i;
    }
  }
  if (elem > min) {
    vec[pi].ind = ind;
    vec[pi].val = elem;
  }
  
  return;
}

__device__ void swap(Elemento *xp, Elemento *yp) {
  //intercambio de elementos
  Elemento temp = *xp;
  *xp = *yp;
  *yp = temp;
}

__device__ void bubbleSort(Elemento *vec, int row, int colTam) {
  //ordenamiento de elementos basico (solo para pocos elementos)
  int i, j;
  for (i = 0; i < colTam - 1; i++)
    for (j = 0; j < colTam - i - 1; j++)
      if (vec[row * colTam + j].val < vec[row * colTam + j + 1].val)
        swap(&vec[row * colTam + j], &vec[row * colTam + j + 1]);
}


__global__ void applyModelKernel(Matrix R, MatrixElemento knn,
  MatrixElemento rec, float *x) {
    
  int u = blockIdx.x * blockDim.x + threadIdx.x;
  if (u < rec.h) {
    for (int i = 0; i < knn.h; i++) {
      x[u * knn.h + i] = 0;
      for (int j = 0; j < knn.w; j++) {
        x[u * knn.h + i] +=
          knn.d[i * knn.w + j].val * R.d[u + knn.d[i * knn.w + j].ind * R.w]; 
      }

      if (R.d[u + i * R.w] != 0)
        x[u * knn.h + i] = 0;
    }

    for (int i = 0; i < rec.w; i++) {
      rec.d[u * rec.w + i].val = 0;
    }

    for (int i = 0; i < knn.h; i++)
      actualizaKValores(&rec.d[u * rec.w], rec.w, x[u * knn.h + i], i);

    bubbleSort(rec.d, u, rec.w);
  }
}


void applyModelAllGPU(Matrix R, MatrixElemento knn,
  MatrixElemento rec) {
  hipError_t err = hipSuccess;

  Matrix d_R;
  d_R.w = R.w; // n = |U|
  d_R.h = R.h; // m = |I|
  size_t size = R.w * R.h * sizeof(float);
  err = hipMalloc(&d_R.d, size);
  err = hipMemcpy(d_R.d, R.d, size, hipMemcpyHostToDevice);

  MatrixElemento d_knn;
  d_knn.w = knn.w;
  d_knn.h = knn.h;
  size = knn.w * knn.h * sizeof(Elemento);
  err = hipMalloc(&d_knn.d, size);
  err = hipMemcpy(d_knn.d, knn.d, size, hipMemcpyHostToDevice);

  MatrixElemento d_rec;
  d_rec.w = rec.w; //nRecomenciones
  d_rec.h = rec.h; //n
  size = rec.w * rec.h * sizeof(Elemento);
  err = hipMalloc(&d_rec.d, size);

  float *d_x;
  hipMalloc(&d_x, sizeof(float) * R.h * R.w);

  applyModelKernel<<<(rec.h + BLOCK_SIZE_IK - 1) / BLOCK_SIZE_IK, BLOCK_SIZE_IK>>>(d_R, d_knn, d_rec, d_x);

  err = hipMemcpy(rec.d, d_rec.d, size, hipMemcpyDeviceToHost);

  if (err != hipSuccess) {
    fprintf(stderr, "falló el copiado de rec (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  hipFree(d_x);
  hipFree(d_rec.d);
  hipFree(d_R.d);
  hipFree(d_knn.d);

  return;
}


